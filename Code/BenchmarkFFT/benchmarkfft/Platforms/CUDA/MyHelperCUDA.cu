#include "hip/hip_runtime.h"
#include "MyHelperCUDA.cuh"

__global__ void twiddle_factors(cpx *W, float angle, int n)
{
    int i = (blockIdx.x * blockDim.x + threadIdx.x);
    SIN_COS_F(angle * i, &W[i].y, &W[i].x);
}

__global__ void bit_reverse(cpx *in, cpx *out, float scale, int lead)
{
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int p = BIT_REVERSE(i, lead);
    out[p].x = in[i].x * scale;
    out[p].y = in[i].y * scale;
}

__global__ void bit_reverse(cpx *x, float dir, int lead, int n)
{
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int p = BIT_REVERSE(i, lead);
    cpx tmp;
    if (i < p) {
        tmp = x[i];
        x[i] = x[p];
        x[p] = tmp;
    }
    if (dir > 0) {
        x[i].x = x[i].x / (float)n;
        x[i].y = x[i].y / (float)n;
    }
}

__global__ void _kernelTranspose(cpx *in, cpx *out, int n)
{
    // Banking issues when TILE_DIM % WARP_SIZE == 0, current WARP_SIZE == 32
    __shared__ cpx tile[TILE_DIM][TILE_DIM + 1];

    // Write to shared from Global (in)
    int x = blockIdx.x * TILE_DIM + threadIdx.x;
    int y = blockIdx.y * TILE_DIM + threadIdx.y;
    for (int j = 0; j < TILE_DIM; j += THREAD_TILE_DIM)
        for (int i = 0; i < TILE_DIM; i += THREAD_TILE_DIM)
            tile[threadIdx.y + j][threadIdx.x + i] = in[(y + j) * n + (x + i)];

    SYNC_THREADS;
    // Write to global
    x = blockIdx.y * TILE_DIM + threadIdx.x;
    y = blockIdx.x * TILE_DIM + threadIdx.y;
    for (int j = 0; j < TILE_DIM; j += THREAD_TILE_DIM)
        for (int i = 0; i < TILE_DIM; i += THREAD_TILE_DIM)
            out[(y + j) * n + (x + i)] = tile[threadIdx.x + i][threadIdx.y + j];
}

__global__ void _kernelTranspose(cuSurf in, cuSurf out, int n)
{
    // Banking issues when TILE_DIM % WARP_SIZE == 0, current WARP_SIZE == 32
    __shared__ cpx tile[TILE_DIM][TILE_DIM + 1];

    // Write to shared from Global (in)
    int x = blockIdx.x * TILE_DIM + threadIdx.x;
    int y = blockIdx.y * TILE_DIM + threadIdx.y;
    for (int j = 0; j < TILE_DIM; j += THREAD_TILE_DIM)
        for (int i = 0; i < TILE_DIM; i += THREAD_TILE_DIM)
            SURF2D_READ(&(tile[threadIdx.y + j][threadIdx.x + i]), in, x + i, y + j);
    //tile[threadIdx.y + j][threadIdx.x + i] = in[(y + j) * n + (x + i)];

    SYNC_THREADS;
    // Write to global
    x = blockIdx.y * TILE_DIM + threadIdx.x;
    y = blockIdx.x * TILE_DIM + threadIdx.y;
    for (int j = 0; j < TILE_DIM; j += THREAD_TILE_DIM)
        for (int i = 0; i < TILE_DIM; i += THREAD_TILE_DIM)
            SURF2D_WRITE(tile[threadIdx.x + i][threadIdx.y + j], out, x + i, y + j);
    //out[(y + j) * n + (x + i)] = tile[threadIdx.x + i][threadIdx.y + j];
}

int checkValidConfig(int blocks, int n)
{
    if (blocks > NO_STREAMING_MULTIPROCESSORS) {
        switch (MAX_BLOCK_SIZE)
        {
        case 256:   return blocks <= 32;    // 2^14
        case 512:   return blocks <= 16;    // 2^14
        case 1024:  return blocks <= 4;     // 2^13
            // Default is a configurable limit, essentially blocksize limits the number of treads that can perform the synchronization.
        default:    return n <= MAX_BLOCK_SIZE * MAX_BLOCK_SIZE;
        }
    }
    return 1;
}

void set_block_and_threads(int *numBlocks, int *threadsPerBlock, int size)
{
    if (size > MAX_BLOCK_SIZE) {
        *numBlocks = size / MAX_BLOCK_SIZE;
        *threadsPerBlock = MAX_BLOCK_SIZE;
    }
    else {
        *numBlocks = 1;
        *threadsPerBlock = size;
    }
}

void set_block_and_threads2D(dim3 *numBlocks, int *threadsPerBlock, int n)
{
    numBlocks->x = n;
    int n2 = n >> 1;
    if (n2 > MAX_BLOCK_SIZE) {
        numBlocks->y = n2 / MAX_BLOCK_SIZE;
        *threadsPerBlock = MAX_BLOCK_SIZE;
    }
    else {
        numBlocks->y = 1;
        *threadsPerBlock = n2;
    }
}

void set_block_and_threads_transpose(dim3 *bTrans, dim3 *tTrans, int n)
{
    bTrans->z = tTrans->z = 1;
    bTrans->x = bTrans->y = (n / TILE_DIM);
    tTrans->x = tTrans->y = THREAD_TILE_DIM;
}

void set2DBlocksNThreads(dim3 *bFFT, dim3 *tFFT, dim3 *bTrans, dim3 *tTrans, int n)
{
    int n2 = n >> 1;
    (*bFFT).x = n;
    (*bFFT).z = (*tFFT).y = (*tFFT).z = (*bTrans).z = (*tTrans).z = 1;
    (*bTrans).x = (*bTrans).y = (n / TILE_DIM);
    (*tTrans).x = (*tTrans).y = THREAD_TILE_DIM;
    if (n2 > MAX_BLOCK_SIZE) {
        (*bFFT).y = n2 / MAX_BLOCK_SIZE;
        (*tFFT).x = MAX_BLOCK_SIZE;
    }
    else {
        (*bFFT).y = 1;
        (*tFFT).x = n2;
    }
}

void checkCudaError(char *msg)
{
    hipError_t e;
    if (e = hipGetLastError()) printf("%s:\n%s: %s\n", msg, hipGetErrorName(e), hipGetErrorString(e));
}

void checkCudaError()
{
    hipError_t e;
    if (e = hipGetLastError()) printf("%s: %s\n", hipGetErrorName(e), hipGetErrorString(e));
}

cpx* read_image(char *name, int *n)
{
    image image;
    color_component *cp;
    FILE *fp;
    fopen_s(&fp, name, "rb");
    image = get_ppm(fp);
    if (!image || image->width != image->height)
        return NULL;

    int size = *n = image->width;
    cpx *seq = (cpx *)malloc(sizeof(cpx) * size * size);
    for (int y = 0; y < (int)image->height; ++y) {
        for (int x = 0; x < (int)image->width; ++x) {
            cp = GET_PIXEL(image, x, y);
            seq[y * size + x] = make_hipComplex((cp[0] + cp[1] + cp[2]) / (3.f * 255.f), 0.f);
        }
    }
    free_img(image);
    return seq;
}

void normalized_cpx_values(cpx* seq, int n, double *min_val, double *range, double *avg)
{
    double min_v = 99999999999;
    double max_v = -99999999999;
    double sum_v = 0.0;
    double tmp = 0.0;
    for (int i = 0; i < n; ++i) {
        tmp = hipCabsf(seq[i]);
        min_v = min(min_v, tmp);
        max_v = max(max_v, tmp);
        sum_v += tmp;
    }
    *min_val = min_v;
    *range = max_v - min_v;
    *avg = sum_v / (double)n;
}

void write_normalized_image(char *name, cpx* seq, int n)
{
    image image;
    FILE  *fp;
    double minVal, range, avg, mag, val;
    normalized_cpx_values(seq, n, &minVal, &range, &avg);
    double avg_pos = 0.4;
    double scale = tan(avg_pos * (M_PI / 2)) / ((avg - minVal) / range);
    image = alloc_img(n, n);
    fopen_s(&fp, name, "wb");
    for (int y = 0; y < n; ++y) {
        for (int x = 0; x < n; ++x) {
            mag = hipCabsf(seq[y * n + x]);
            val = ((mag - minVal) / range);
            val = (atan(val * scale) / (M_PI / 2.0)) * 255.0;
            color_component col = (unsigned char)(val > 255.0 ? 255 : val);
            put_pixel_unsafe(image, x, y, col, col, col);
        }
    }
    output_ppm(fp, image);
    fclose(fp);
    free_img(image);
}

void normalized_image(cpx* seq, int n)
{
    double minVal, range, avg, mag, val;
    normalized_cpx_values(seq, n, &minVal, &range, &avg);
    double avg_pos = 0.8;
    double scale = tan(avg_pos * (M_PI / 2)) / ((avg - minVal) / range);
    for (int y = 0; y < n; ++y) {
        for (int x = 0; x < n; ++x) {
            mag = hipCabsf(seq[y * n + x]);
            val = ((mag - minVal) / range);
            val = (atan(val * scale) / (M_PI / 2.0));
            seq[y * n + x] = make_hipFloatComplex((float)(val > 1.0 ? 1 : val), 0.f);
        }
    }
}

void write_image(char *name, char *type, cpx* seq, int n)
{
    image image;
    FILE  *fp;
    image = alloc_img(n, n);
    char filename[50];
    sprintf_s(filename, 50, "out/img/%s_%u_%s.ppm", name, n, type);
    fopen_s(&fp, filename, "wb");
    for (int y = 0; y < n; ++y) {
        for (int x = 0; x < n; ++x) {
            color_component val = (unsigned char)((seq[y * n + x].x) * 255.f);
            put_pixel_unsafe(image, x, y, val, val, val);
        }
    }
    output_ppm(fp, image);
    fclose(fp);
    free_img(image);
}

void clear_image(cpx* seq, int n)
{
    for (int i = 0; i < n; ++i)
        seq[i] = make_hipFloatComplex(1.f, 1.f);
}

void cpPixel(int px, int px2, cpx *in, cpx *out)
{
    int p, p2;
    p = px * 3;
    p2 = px2 * 3;
    out[p] = in[p2];
    out[p + 1] = in[p2 + 1];
    out[p + 2] = in[p2 + 2];
}

cpx* fftShift(cpx *seq, int n)
{
    cpx *out = (cpx *)malloc(sizeof(cpx)*n*n);
    int px1, px2;
    int n2 = n / 2;
    for (int y = 0; y < n2; ++y) {
        for (int x = 0; x < n2; ++x) {
            px1 = y * n + x;
            px2 = (y + n2) * n + (x + n2);
            out[px1] = seq[px2];
            out[px2] = seq[px1];
        }
    }
    for (int y = 0; y < n2; ++y) {
        for (int x = n2; x < n; ++x) {
            px1 = y * n + x;
            px2 = (y + n2) * n + (x - n2);
            out[px1] = seq[px2];
            out[px2] = seq[px1];
        }
    }
    return out;
}

#define ERROR_MARGIN 0.0001

static LARGE_INTEGER StartingTime, EndingTime, ElapsedMicroseconds, Frequency;

void startTimer()
{
    QueryPerformanceFrequency(&Frequency);
    QueryPerformanceCounter(&StartingTime);
}

double stopTimer()
{
    QueryPerformanceCounter(&EndingTime);
    ElapsedMicroseconds.QuadPart = EndingTime.QuadPart - StartingTime.QuadPart;
    ElapsedMicroseconds.QuadPart *= 1000000;
    ElapsedMicroseconds.QuadPart /= Frequency.QuadPart;
    return (double)ElapsedMicroseconds.QuadPart;
}

// Useful functions for debugging
void console_print(cpx *seq, int n)
{
    for (int i = 0; i < n; ++i) printf("%f\t%f\n", seq[i].x, seq[i].y);
}

void console_print_cpx_img(cpx *seq, int n)
{
    printf("\n");
    for (int y = 0; y < n; ++y) {
        for (int x = 0; x < n; ++x) {
            printf("%.2f\t", seq[y * n + x].x);
        }
        printf("\n");
    }
    printf("\n");
    for (int y = 0; y < n; ++y) {
        for (int x = 0; x < n; ++x) {
            printf("%.2f\t", seq[y * n + x].y);
        }
        printf("\n");
    }
}

unsigned int power(unsigned int base, unsigned int exp)
{
    if (exp == 0)
        return 1;
    unsigned int value = base;
    for (unsigned int i = 1; i < exp; ++i) {
        value *= base;
    }
    return value;
}

unsigned int power2(unsigned int exp)
{
    return power(2, exp);
}

int checkError(cpx *seq, cpx *ref, float refScale, int n, int print)
{
    int j;
    double re, im, i_val, r_val;
    re = im = 0.0;
    for (j = 0; j < n; ++j) {
        r_val = abs(refScale * seq[j].x - ref[j].x);
        i_val = abs(refScale * seq[j].y - ref[j].y);
        re = re > r_val ? re : r_val;
        im = im > i_val ? im : i_val;
    }
    if (print == 1) printf("Error\tre(e): %f\t im(e): %f\t@%u\n", re, im, n);
    return re > ERROR_MARGIN || im > ERROR_MARGIN;
}

int checkError(cpx *seq, cpx *ref, int n, int print)
{
    return checkError(seq, ref, 1.f, n, print);
}

int checkError(cpx *seq, cpx *ref, int n)
{
    return checkError(seq, ref, n, 0);
}

cpx *get_seq(int n)
{
    return get_seq(n, 0);
}

cpx *get_seq(int n, int sinus)
{
    int i;
    cpx *seq;
    seq = (cpx *)malloc(sizeof(cpx) * n);
    for (i = 0; i < n; ++i) {
        seq[i].x = sinus == 0 ? 0.f : (float)sin(M_2_PI * (((double)i) / n));
        seq[i].y = 0.f;
    }
    return seq;
}

cpx *get_seq(int n, cpx *src)
{
    int i;
    cpx *seq;
    seq = (cpx *)malloc(sizeof(cpx) * n);
    for (i = 0; i < n; ++i) {
        seq[i].x = src[i].x;
        seq[i].y = src[i].y;
    }
    return seq;
}

cpx *get_sin_img(int n)
{
    cpx *seq;
    seq = (cpx *)malloc(sizeof(cpx) * n * n);
    for (int y = 0; y < n; ++y)
        for (int x = 0; x < n; ++x)
            seq[y * n + x] = make_hipFloatComplex((float)sin(M_2_PI * (((double)x) / n)), 0.f);
    return seq;
}

int cmp(const void *x, const void *y)
{
    double xx = *(double*)x, yy = *(double*)y;
    if (xx < yy) return -1;
    if (xx > yy) return  1;
    return 0;
}

double avg(double m[], int n)
{
    int i, cnt, end;
    double sum;
    qsort(m, n, sizeof(double), cmp);
    sum = 0.0;
    cnt = 0;
    end = n < 5 ? n - 1 : 5;
    for (i = 0; i < end; ++i) {
        sum += m[i];
        ++cnt;
    }
    return (sum / (double)cnt);
}

void _cudaMalloc(int n, cpx **dev_in, cpx **dev_out, cpx **dev_W)
{
    *dev_in = 0;
    *dev_out = 0;
    hipMalloc((void**)dev_in, n * sizeof(cpx));
    hipMalloc((void**)dev_out, n * sizeof(cpx));
    if (dev_W != NULL) {
        *dev_W = 0;
        hipMalloc((void**)dev_W, (n / 2) * sizeof(cpx));
    }
}

void _fftTestSeq(int n, cpx **in, cpx **ref, cpx **out)
{
    *in = get_seq(n, 1);
    *ref = get_seq(n, *in);
    *out = get_seq(n);
}

void fftMalloc(int n, cpx **dev_in, cpx **dev_out, cpx **dev_W, cpx **in, cpx **ref, cpx **out)
{
    _cudaMalloc(n, dev_in, dev_out, dev_W);
    if (in == NULL && ref == NULL && out == NULL)
        return;
    _fftTestSeq(n, in, ref, out);
}

void _cudaFree(cpx **dev_in, cpx **dev_out, cpx **dev_W)
{
    hipFree(*dev_in);
    hipFree(*dev_out);
    if (dev_W != NULL) hipFree(*dev_W);
}

void _fftFreeSeq(cpx **in, cpx **ref, cpx **out)
{
    free(*in);
    free(*ref);
    free(*out);
}

int fftResultAndFree(int n, cpx **dev_in, cpx **dev_out, cpx **dev_W, cpx **in, cpx **ref, cpx **out)
{
    int result;
    _cudaFree(dev_in, dev_out, dev_W);
    hipDeviceSynchronize();
    if (in == NULL && ref == NULL && out == NULL)
        return 0;
    result = checkError(*in, *ref, n);
    _fftFreeSeq(in, out, ref);
    hipError_t cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }
    return result;
}

void fft2DSetup(cpx **in, cpx **ref, cpx **dev_i, cpx **dev_o, size_t *size, char *image_name, int sinus, int n)
{
    if (sinus) {
        *in = get_sin_img(n);
        *ref = get_sin_img(n);
    }
    else {
        char input_file[40];
        sprintf_s(input_file, 40, "%s/%u.ppm", image_name, n);
        int sz;
        *in = read_image(input_file, &sz);
        *ref = read_image(input_file, &sz);
    }
    *size = n * n * sizeof(cpx);
    hipMalloc((void**)dev_i, *size);
    hipMalloc((void**)dev_o, *size);
    hipMemcpy(*dev_i, *in, *size, hipMemcpyHostToDevice);
}

void fft2DShakedown(cpx **in, cpx **ref, cpx **dev_i, cpx **dev_o)
{
    free(*in);
    free(*ref);
    hipFree(*dev_i);
    hipFree(*dev_o);
    hipError_t cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
    }
}

int fft2DCompare(cpx *in, cpx *ref, cpx *dev, size_t size, int len)
{
    hipMemcpy(in, dev, size, hipMemcpyDeviceToHost);
    for (int i = 0; i < len; ++i) {
        if (hipCabsf(hipCsubf(in[i], ref[i])) > 0.0001) {
            return 0;
        }
    }
    return 1;
}

int fft2DCompare(cpx *in, cpx *ref, cpx *dev, size_t size, int len, double *relDiff)
{
    double mDiff = 0.0;
    double mVal = -1;
    hipMemcpy(in, dev, size, hipMemcpyDeviceToHost);
    for (int i = 0; i < len; ++i) {
        mVal = max(mVal, max(hipCabsf(in[i]), hipCabsf(ref[i])));
        double tmp = hipCabsf(hipCsubf(in[i], ref[i]));
        mDiff = tmp > mDiff ? tmp : mDiff;
    }
    *relDiff = (mDiff / mVal);
    return *relDiff < 0.00001;
}

void cudaCheckError(hipError_t err)
{
    if (err) {
        printf("\n%s: %s\n", hipGetErrorName(err), hipGetErrorString(err));
        getchar();
        exit(err);
    }
}

void cudaCheckError()
{
    cudaCheckError(hipGetLastError());
}

void fft2DSurfSetup(cpx **in, cpx **ref, size_t *size, char *image_name, int sinus, int n, hipArray **cuInputArray, hipArray **cuOutputArray, cuSurf *inputSurfObj, cuSurf *outputSurfObj)
{
    if (sinus) {
        *in = get_sin_img(n);
        *ref = get_sin_img(n);
    }
    else {
        char input_file[40];
        sprintf_s(input_file, 40, "%s/%u.ppm", image_name, n);
        int sz;
        *in = read_image(input_file, &sz);
        *ref = read_image(input_file, &sz);
    }
    *size = n * n * sizeof(cpx);
    // Allocate CUDA arrays in device memory
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float2>();
    hipMallocArray(cuInputArray, &channelDesc, n, n, hipArraySurfaceLoadStore);
    cudaCheckError();
    if (cuOutputArray != NULL) {
        hipMallocArray(cuOutputArray, &channelDesc, n, n, hipArraySurfaceLoadStore);
    }
    // Specify surface
    struct hipResourceDesc resDesc;
    memset(&resDesc, 0, sizeof(resDesc));
    resDesc.resType = hipResourceTypeArray;
    // Create the surface objects
    resDesc.res.array.array = *cuInputArray;
    *inputSurfObj = 0;
    hipCreateSurfaceObject(inputSurfObj, &resDesc);
    cudaCheckError();
    if (outputSurfObj != NULL) {
        resDesc.res.array.array = *cuOutputArray;
        *outputSurfObj = 0;
        hipCreateSurfaceObject(outputSurfObj, &resDesc);
        cudaCheckError();
    }
}