#include "hip/hip_runtime.h"
#include "cuda_fft.cuh"
#if defined(_NVIDIA)
// Comment: Max values all over seems to give the biggest benefit when running large sets of data
//          Smaller sets benefits a little from smaller groups/blocks.
//          I my opinion, the large values seems to suit CUDA best.

#define CU_BLOCK_SIZE 1024
#define CU_TILE_DIM 64 // Sets local/shared mem when transposing
#define CU_BLOCK_DIM 32 // Sets threads when transposing

__host__ void cuda_fft(transform_direction dir, cpx *in, cpx *out, int n, bool experimental);

__global__ void cuda_kernel_global(cpx *in, float global_angle, unsigned int lmask, int steps, int dist);
__global__ void cuda_kernel_global_row(cpx *in, float global_angle, unsigned int lmask, int steps, int dist);

__global__ void cuda_kernel_local(cpx *in, cpx *out, float local_angle, int steps_left, int leading_bits, float scalar);
__global__ void cuda_kernel_local_row(cpx *in, cpx *out, float local_angle, int steps_left, int leading_bits, float scalar);

__global__ void cuda_transpose_kernel(cpx *in, cpx *out, int n);

// -------------------------------
//
// Testing
//
// -------------------------------

#include "../../Common/cpx_debug.h"

__host__ int cuda_validate(int n)
{
    cpx *in, *ref, *out, *dev_in, *dev_out;
    size_t buffer_size = batch_size(n) * sizeof(cpx);
    cuda_setup_buffers(n, &dev_in, &dev_out, &in, &ref, &out);
    hipMemcpy(dev_in, in, buffer_size, hipMemcpyHostToDevice);

    cuda_fft(FFT_FORWARD, dev_in, dev_out, n);
    hipDeviceSynchronize();
    hipMemcpy(out, dev_out, buffer_size, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    double diff = diff_forward_sinus(out, batch_count(n), n);
    
    cuda_fft(FFT_INVERSE, dev_out, dev_in, n);
    hipDeviceSynchronize();
    hipMemcpy(in, dev_in, buffer_size, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    hipMemcpy(out, dev_out, buffer_size, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    return (cuda_shakedown(n, &dev_in, &dev_out, &in, &ref, &out) != 1) && (diff <= RELATIVE_ERROR_MARGIN);
}

__host__ int cuda_2d_validate(int n, bool write_img)
{
    cpx *host_buffer, *ref, *dev_in, *dev_out;
    size_t size;
    cuda_setup_buffers_2d(&host_buffer, &ref, &dev_in, &dev_out, &size, n);
    hipMemcpy(dev_in, host_buffer, size, hipMemcpyHostToDevice);
    cuda_fft_2d(FFT_FORWARD, &dev_in, &dev_out, n);
    hipDeviceSynchronize();

    if (write_img) {
        hipMemcpy(host_buffer, dev_out, size, hipMemcpyDeviceToHost);
        write_normalized_image("CUDA", "freq", host_buffer, n, true);
    }
    cuda_fft_2d(FFT_INVERSE, &dev_out, &dev_in, n);
    hipDeviceSynchronize();

    if (write_img) {
        hipMemcpy(host_buffer, dev_in, size, hipMemcpyDeviceToHost);
        write_image("CUDA", "spat", host_buffer, n);
    }
    int res = cuda_compare_result(host_buffer, ref, dev_in, size, batch_size(n * n));
    cuda_shakedown_2d(&host_buffer, &ref, &dev_in, &dev_out);
    return res;
}

__host__ double cuda_performance(int n)
{
    double measures[64];
    cpx *in, *ref, *out, *dev_in, *dev_out;

    cuda_setup_buffers(n, &dev_in, &dev_out, &in, &ref, &out);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float milliseconds = 0;
    hipMemcpy(dev_in, in, n * sizeof(cpx), hipMemcpyHostToDevice);
    for (int i = 0; i < number_of_tests; ++i) {
        hipDeviceSynchronize();
        hipEventRecord(start);
        cuda_fft(FFT_FORWARD, dev_in, dev_out, n);
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&milliseconds, start, stop);
        measures[i] = milliseconds * 1000;
    }
    cuda_shakedown(n, &dev_in, &dev_out, &in, &ref, &out);
    double t = average_best(measures, number_of_tests);
    return t;
}

__host__ double cuda_2d_performance(int n)
{
    double measures[64];
    cpx *in, *ref, *dev_in, *dev_out;
    size_t size;
    cuda_setup_buffers_2d(&in, &ref, &dev_in, &dev_out, &size, n);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float milliseconds = 0;
    for (int i = 0; i < number_of_tests; ++i) {
        hipDeviceSynchronize();
        hipEventRecord(start);
        cuda_fft_2d(FFT_FORWARD, &dev_in, &dev_out, n);
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&milliseconds, start, stop);
        measures[i] = milliseconds * 1000;
    }
    cuda_shakedown_2d(&in, &ref, &dev_in, &dev_out);
    return average_best(measures, number_of_tests);
}

// -------------------------------
//
// Algorithm
//
// -------------------------------

__host__ __inline void set_block_and_threads(dim3 *number_of_blocks, int *threads_per_block, const int block_size, const bool dim2, const int n)
{
    const int n_half = n >> 1;
    const bool multi_blocks = (n_half > block_size);
    *threads_per_block = multi_blocks ? block_size : n_half;
    number_of_blocks->x = dim2 ? n : multi_blocks ? n_half / block_size : 1;
    number_of_blocks->y = dim2 ? multi_blocks ? n_half / block_size : 1 : number_of_blocks->x;
}

__host__ void cuda_fft(transform_direction dir, cpx *in, cpx *out, int n)
{
    fft_args args;
    dim3 blocks;
    int threads;
    set_block_and_threads(&blocks, &threads, CU_BLOCK_SIZE, (n >> 1));
    set_fft_arguments(&args, dir, blocks.y, CU_BLOCK_SIZE, n);
    hipDeviceSetCacheConfig(hipFuncCachePreferL1);
    if (blocks.y > 1) {
        while (--args.steps_left > args.steps_gpu) {
            cuda_kernel_global KERNEL_ARGS2(blocks, threads)(in, args.global_angle, 0xFFFFFFFF << args.steps_left, args.steps++, args.dist >>= 1);
        }
        ++args.steps_left;
    }
    cuda_kernel_local KERNEL_ARGS3(blocks, threads, sizeof(cpx) * args.n_per_block) (in, out, args.local_angle, args.steps_left, args.leading_bits, args.scalar);
}

__host__ __inline void cuda_fft_2d_helper(transform_direction dir, cpx *dev_in, cpx *dev_out, int n)
{
    fft_args args;
    dim3 blocks;
    int threads;
    set_block_and_threads_2d(&blocks, &threads, CU_BLOCK_SIZE, n);
    set_fft_arguments(&args, dir, blocks.y, CU_BLOCK_SIZE, n);
    if (blocks.y > 1) {
        while (--args.steps_left > args.steps_gpu) {
            cuda_kernel_global_row KERNEL_ARGS2(blocks, threads)(dev_in, args.global_angle, 0xFFFFFFFF << args.steps_left, args.steps++, args.dist >>= 1);
        }
        ++args.steps_left;
    }
    cuda_kernel_local_row KERNEL_ARGS3(blocks, threads, sizeof(cpx) * args.n_per_block) (dev_in, dev_out, args.local_angle, args.steps_left, args.leading_bits, args.scalar);
}

__host__ void cuda_fft_2d(transform_direction dir, cpx **dev_in, cpx **dev_out, int n)
{
    dim3 blocks;
    dim3 threads;
    set_block_and_threads_transpose(&blocks, &threads, CU_TILE_DIM, CU_BLOCK_DIM, n);
    cuda_fft_2d_helper(dir, *dev_in, *dev_out, n);
    cuda_transpose_kernel KERNEL_ARGS2(blocks, threads) (*dev_out, *dev_in, n);
    cuda_fft_2d_helper(dir, *dev_in, *dev_out, n);
    cuda_transpose_kernel KERNEL_ARGS2(blocks, threads) (*dev_out, *dev_in, n);
    swap_buffer(dev_in, dev_out);
}

// -------------------------------
//
// Device
//
// -------------------------------

#define CU_BATCH_ID (blockIdx.x)
#define CU_N_POINTS ((gridDim.y * blockDim.x) << 1)
#define CU_THREAD_ID (blockIdx.y * blockDim.x + threadIdx.x)
#define CU_BLOCK_OFFSET (blockIdx.y * (blockDim.x << 1))

#define CU_CU_BATCH_ID_2D (blockIdx.z)
#define CU_CU_N_POINTS_2D (gridDim.x * gridDim.x)
#define CU_OFFSET_2D ((blockIdx.x + blockIdx.z * gridDim.x) * gridDim.x)
#define CU_COL_ID (blockIdx.y * blockDim.x + threadIdx.x)

#define CU_IMG_DIST (blockIdx.z * gridDim.x * gridDim.x * CU_TILE_DIM * CU_TILE_DIM)

__device__ __inline void cu_contant_geometry(cpx *shared, cpx *in_l, cpx *in_h, float angle, int steps_limit)
{
    cpx w, l, h;
    cpx *out_i = shared + (threadIdx.x << 1),
        *out_ii = out_i + 1;
    float x, y;
    for (int steps = 0; steps < steps_limit; ++steps) {
        l = *in_l;
        h = *in_h;
        x = l.x - h.x;
        y = l.y - h.y;
        SIN_COS_F(angle * (threadIdx.x & (0xFFFFFFFF << steps)), &w.y, &w.x);
        SYNC_THREADS;
        *out_i = { l.x + h.x, l.y + h.y };
        *out_ii = { (w.x * x) - (w.y * y), (w.y * x) + (w.x * y) };
        SYNC_THREADS;
    }
}

__device__ __inline void cuda_partial(cpx *in, cpx *out, cpx *shared, unsigned int in_high, unsigned int offset, float local_angle, int steps_left, int leading_bits, float scalar)
{
    cpx *in_l = shared + threadIdx.x,
        *in_u = shared + in_high;
    *in_l = in[threadIdx.x];
    *in_u = in[in_high];
    cu_contant_geometry(shared, in_l, in_u, local_angle, steps_left);
    out[BIT_REVERSE(threadIdx.x + offset, leading_bits)] = { in_l->x * scalar, in_l->y * scalar };
    out[BIT_REVERSE(in_high + offset, leading_bits)] = { in_u->x * scalar, in_u->y * scalar };
}

__global__ void cuda_kernel_local(cpx *in, cpx *out, float local_angle, int steps_left, int leading_bits, float scalar)
{
    extern __shared__ cpx shared[];
    cuda_partial(in + CU_BLOCK_OFFSET + CU_BATCH_ID * CU_N_POINTS, out + CU_BATCH_ID * CU_N_POINTS, shared, threadIdx.x + blockDim.x, CU_BLOCK_OFFSET, local_angle, steps_left, leading_bits, scalar);
}

__global__ void cuda_kernel_local_row(cpx *in, cpx *out, float local_angle, int steps_left, int leading_bits, float scalar)
{
    extern __shared__ cpx shared[];
    int row_start = gridDim.x * blockIdx.x + CU_CU_BATCH_ID_2D * CU_CU_N_POINTS_2D;
    int offset = (blockIdx.y * blockDim.x) << 1;
    cuda_partial(in + row_start + offset, out + row_start, shared, blockDim.x + threadIdx.x, offset, local_angle, steps_left, leading_bits, scalar);
}

__device__ __inline void cu_global(cpx *in, int tid, float angle, int steps, int dist)
{
    cpx w;
    SIN_COS_F(angle * ((tid << steps) & ((dist - 1) << steps)), &w.y, &w.x);
    cpx l = *in;
    cpx h = in[dist];
    float x = l.x - h.x;
    float y = l.y - h.y;
    *in = { l.x + h.x, l.y + h.y };
    in[dist] = { (w.x * x) - (w.y * y), (w.y * x) + (w.x * y) };
}

__global__ void cuda_kernel_global(cpx *in, float angle, unsigned int lmask, int steps, int dist)
{
    cu_global(in + CU_THREAD_ID + (CU_THREAD_ID & lmask) + CU_BATCH_ID * CU_N_POINTS, CU_THREAD_ID, angle, steps, dist);
}

__global__ void cuda_kernel_global_row(cpx *in, float angle, unsigned int lmask, int steps, int dist)
{
    cu_global(in + (CU_COL_ID + (CU_COL_ID & lmask)) + CU_OFFSET_2D, CU_COL_ID, angle, steps, dist);
}

__global__ void cuda_transpose_kernel(cpx *in, cpx *out, int n)
{
    // Banking issues when CU_TILE_DIM % WARP_SIZE == 0, current WARP_SIZE == 32
    __shared__ cpx tile[CU_TILE_DIM][CU_TILE_DIM + 1];

    // Image offset
    in += CU_IMG_DIST;
    out += CU_IMG_DIST;

    // Write to shared from Global (in)
    int x = blockIdx.x * CU_TILE_DIM + threadIdx.x;
    int y = blockIdx.y * CU_TILE_DIM + threadIdx.y;
#pragma unroll
    for (int j = 0; j < CU_TILE_DIM; j += CU_BLOCK_DIM)
        for (int i = 0; i < CU_TILE_DIM; i += CU_BLOCK_DIM)
            tile[threadIdx.y + j][threadIdx.x + i] = in[(y + j) * n + (x + i)];

    SYNC_THREADS;
    // Write to global
    x = blockIdx.y * CU_TILE_DIM + threadIdx.x;
    y = blockIdx.x * CU_TILE_DIM + threadIdx.y;
#pragma unroll
    for (int j = 0; j < CU_TILE_DIM; j += CU_BLOCK_DIM)
        for (int i = 0; i < CU_TILE_DIM; i += CU_BLOCK_DIM)
            out[(y + j) * n + (x + i)] = tile[threadIdx.x + i][threadIdx.y + j];
}
#endif