#include "hip/hip_runtime.h"
#include "cuda_fft.cuh"
#if defined(_NVIDIA)
// Comment: Max values all over seems to give the biggest benefit when running large sets of data
//          Smaller sets benefits a little from smaller groups/blocks.
//          I my opinion, the large values seems to suit CUDA best.

#define CU_BLOCK_SIZE 1024
#define CU_TILE_DIM 64 // Sets local/shared mem when transposing
#define CU_BLOCK_DIM 32 // Sets threads when transposing

__global__ void cuda_kernel_global    (cpx *in, float global_angle, unsigned int lmask, int steps, int dist);
__global__ void cuda_kernel_global_row(cpx *in, float global_angle, unsigned int lmask, int steps, int dist);

__global__ void cuda_kernel_local    (cpx *in, cpx *out, float local_angle, int steps_left, int leading_bits, float scalar);
__global__ void cuda_kernel_local_row(cpx *in, cpx *out, float local_angle, int steps_left, int leading_bits, float scalar);

__global__ void cuda_transpose_kernel(cpx *in, cpx *out, int n);

// -------------------------------
//
// Testing
//
// -------------------------------

#include "../../Common/cpx_debug.h"

__host__ int cuda_validate(int n)
{
    cpx *in, *ref, *out, *dev_in, *dev_out;
    size_t buffer_size = batch_size(n) * sizeof(cpx);
    cuda_setup_buffers(n, &dev_in, &dev_out, &in, &ref, &out);
    hipMemcpy(dev_in, in, buffer_size, hipMemcpyHostToDevice);

    cuda_fft(FFT_FORWARD, dev_in, dev_out, n);
    hipDeviceSynchronize();
    hipMemcpy(out, dev_out, buffer_size, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    double diff = diff_forward_sinus(out, batch_count(n), n);
    
    cuda_fft(FFT_INVERSE, dev_out, dev_in, n);
    hipDeviceSynchronize();
    hipMemcpy(in, dev_in, buffer_size, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    hipMemcpy(out, dev_out, buffer_size, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    return (cuda_shakedown(n, &dev_in, &dev_out, &in, &ref, &out) != 1) && (diff <= RELATIVE_ERROR_MARGIN);
}

__host__ int cuda_2d_validate(int n, bool write_img)
{
    cpx *host_buffer, *ref, *dev_in, *dev_out;
    size_t size;
    cuda_setup_buffers_2d(&host_buffer, &ref, &dev_in, &dev_out, &size, n);
    hipMemcpy(dev_in, host_buffer, size, hipMemcpyHostToDevice);
    cuda_fft_2d(FFT_FORWARD, &dev_in, &dev_out, n);
    hipDeviceSynchronize();

    if (write_img) {
        hipMemcpy(host_buffer, dev_out, size, hipMemcpyDeviceToHost);
        write_normalized_image("CUDA", "freq", host_buffer, n, true);
    }
    cuda_fft_2d(FFT_INVERSE, &dev_out, &dev_in, n);
    hipDeviceSynchronize();

    if (write_img) {
        hipMemcpy(host_buffer, dev_in, size, hipMemcpyDeviceToHost);
        write_image("CUDA", "spat", host_buffer, n);
    }
    int res = cuda_compare_result(host_buffer, ref, dev_in, size, batch_size(n * n));
    cuda_shakedown_2d(&host_buffer, &ref, &dev_in, &dev_out);
    return res;
}

__host__ double cuda_performance(int n)
{
    double measures[64];
    cpx *in, *ref, *out, *dev_in, *dev_out;

    cuda_setup_buffers(n, &dev_in, &dev_out, &in, &ref, &out);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float milliseconds = 0;
    hipMemcpy(dev_in, in, n * sizeof(cpx), hipMemcpyHostToDevice);
    for (int i = 0; i < number_of_tests; ++i) {
        hipDeviceSynchronize();
        hipEventRecord(start);
        cuda_fft(FFT_FORWARD, dev_in, dev_out, n);
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&milliseconds, start, stop);
        measures[i] = milliseconds * 1000;
    }
    cuda_shakedown(n, &dev_in, &dev_out, &in, &ref, &out);
    double t = average_best(measures, number_of_tests);
    return t;
}

__host__ double cuda_2d_performance(int n)
{
    double measures[64];
    cpx *in, *ref, *dev_in, *dev_out;
    size_t size;
    cuda_setup_buffers_2d(&in, &ref, &dev_in, &dev_out, &size, n);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float milliseconds = 0;
    for (int i = 0; i < number_of_tests; ++i) {
        hipDeviceSynchronize();
        hipEventRecord(start);
        cuda_fft_2d(FFT_FORWARD, &dev_in, &dev_out, n);
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&milliseconds, start, stop);
        measures[i] = milliseconds * 1000;
    }
    cuda_shakedown_2d(&in, &ref, &dev_in, &dev_out);
    return average_best(measures, number_of_tests);
}

// -------------------------------
//
// Algorithm
//
// -------------------------------

__host__ __inline void set_block_and_threads(dim3 *number_of_blocks, int *threads_per_block, const int block_size, const bool dim2, const int n)
{
    const int n_half = n >> 1;
    const bool multi_blocks = (n_half > block_size);
    *threads_per_block = multi_blocks ? block_size : n_half;
    number_of_blocks->x = dim2 ? n : multi_blocks ? n_half / block_size : 1;
    number_of_blocks->y = dim2 ? multi_blocks ? n_half / block_size : 1 : number_of_blocks->x;
}

__host__ void cuda_fft(transform_direction dir, cpx *in, cpx *out, int n)
{
    fft_args args;
    dim3 blocks;
    int threads;
    set_block_and_threads(&blocks, &threads, CU_BLOCK_SIZE, (n >> 1));
    set_fft_arguments(&args, dir, blocks.y, CU_BLOCK_SIZE, n);
    hipDeviceSetCacheConfig(hipFuncCachePreferL1);
    if (blocks.y > 1) {
        while (--args.steps_left > args.steps_gpu) {
            cuda_kernel_global KERNEL_ARGS2(blocks, threads)(in, args.global_angle, 0xFFFFFFFF << args.steps_left, args.steps++, args.dist >>= 1);
        }
        ++args.steps_left;
    }
    cuda_kernel_local KERNEL_ARGS3(blocks, threads, sizeof(cpx) * args.n_per_block) (in, out, args.local_angle, args.steps_left, args.leading_bits, args.scalar);
}

__host__ __inline void cuda_fft_2d_helper(transform_direction dir, cpx *dev_in, cpx *dev_out, int n)
{
    fft_args args;
    dim3 blocks;
    int threads;
    set_block_and_threads_2d(&blocks, &threads, CU_BLOCK_SIZE, n);
    set_fft_arguments(&args, dir, blocks.y, CU_BLOCK_SIZE, n);
    if (blocks.y > 1) {
        while (--args.steps_left > args.steps_gpu) {
            cuda_kernel_global_row KERNEL_ARGS2(blocks, threads)(dev_in, args.global_angle, 0xFFFFFFFF << args.steps_left, args.steps++, args.dist >>= 1);
        }
        ++args.steps_left;
    }
    cuda_kernel_local_row KERNEL_ARGS3(blocks, threads, sizeof(cpx) * args.n_per_block) (dev_in, dev_out, args.local_angle, args.steps_left, args.leading_bits, args.scalar);
}

__host__ void cuda_fft_2d(transform_direction dir, cpx **dev_in, cpx **dev_out, int n)
{
    dim3 blocks;
    dim3 threads;
    set_block_and_threads_transpose(&blocks, &threads, CU_TILE_DIM, CU_BLOCK_DIM, n);
    cuda_fft_2d_helper(dir, *dev_in, *dev_out, n);
    cuda_transpose_kernel KERNEL_ARGS2(blocks, threads) (*dev_out, *dev_in, n);
    cuda_fft_2d_helper(dir, *dev_in, *dev_out, n);
    cuda_transpose_kernel KERNEL_ARGS2(blocks, threads) (*dev_out, *dev_in, n);
    swap_buffer(dev_in, dev_out);
}

// -------------------------------
//
// Device
//
// -------------------------------

__device__ __inline void cu_contant_geometry(cpx *shared, cpx *in_l, cpx *in_h, float angle, int steps_limit)
{
    cpx w, l, h;
    cpx *out_i = shared + (threadIdx.x << 1),
        *out_ii = out_i + 1;
    float x, y;
    for (int steps = 0; steps < steps_limit; ++steps) {
        l = *in_l;
        h = *in_h;
        x = l.x - h.x;
        y = l.y - h.y;
        SIN_COS_F(angle * (threadIdx.x & (0xFFFFFFFF << steps)), &w.y, &w.x);
        SYNC_THREADS;
        *out_i = { l.x + h.x, l.y + h.y };
        *out_ii = { (w.x * x) - (w.y * y), (w.y * x) + (w.x * y) };
        SYNC_THREADS;
    }
}

__device__ __inline void cuda_partial(cpx *in, cpx *out, cpx *shared, unsigned int in_high, unsigned int offset, float local_angle, int steps_left, int leading_bits, float scalar)
{
    int in_low = threadIdx.x;
    cpx *in_l = shared + in_low,
        *in_u = shared + in_high;
    *in_l = in[in_low];
    *in_u = in[in_high];
    cu_contant_geometry(shared, in_l, in_u, local_angle, steps_left);
    out[BIT_REVERSE(in_low + offset, leading_bits)] = { in_l->x * scalar, in_l->y * scalar };
    out[BIT_REVERSE(in_high + offset, leading_bits)] = { in_u->x * scalar, in_u->y * scalar };
}

__global__ void cuda_kernel_local(cpx *in, cpx *out, float local_angle, int steps_left, int leading_bits, float scalar)
{
    extern __shared__ cpx shared[];    
    int n_block (blockDim.x << 1),
        arg4 (blockIdx.y * n_block),
        arg1 (gridDim.y * n_block * blockIdx.x),
        arg0 (arg1 + arg4),
        arg3 (threadIdx.x + blockDim.x);
    cuda_partial(in + arg0, out + arg1, shared, arg3, arg4, local_angle, steps_left, leading_bits, scalar);
}

__global__ void cuda_kernel_local_row(cpx *in, cpx *out, float local_angle, int steps_left, int leading_bits, float scalar)
{
    extern __shared__ cpx shared[];
    int arg4((blockIdx.y * blockDim.x) << 1),
        arg1((blockIdx.x + blockIdx.z * gridDim.x) * gridDim.x),
        arg0(arg1 + arg4),
        arg3(blockDim.x + threadIdx.x);
    cuda_partial(in + arg0, out + arg1, shared, arg3, arg4, local_angle, steps_left, leading_bits, scalar);
}

__device__ __inline void cu_global(cpx *in, int tid, float angle, int steps, int dist)
{
    cpx w;
    SIN_COS_F(angle * ((tid << steps) & ((dist - 1) << steps)), &w.y, &w.x);
    cpx l = *in;
    cpx h = in[dist];
    float x = l.x - h.x;
    float y = l.y - h.y;
    *in = { l.x + h.x, l.y + h.y };
    in[dist] = { (w.x * x) - (w.y * y), (w.y * x) + (w.x * y) };
}

__global__ void cuda_kernel_global(cpx *in, float angle, unsigned int lmask, int steps, int dist)
{
    int arg1(blockIdx.y * blockDim.x + threadIdx.x),
        arg0(arg1 + (arg1 & lmask) + blockIdx.x * ((gridDim.y * blockDim.x) << 1));
    cu_global(in + arg0, arg1, angle, steps, dist);
}

__global__ void cuda_kernel_global_row(cpx *in, float angle, unsigned int lmask, int steps, int dist)
{
    int arg1(blockIdx.y * blockDim.x + threadIdx.x),
        arg0(arg1 + (arg1 & lmask) + (blockIdx.x + blockIdx.z * gridDim.x) * gridDim.x);
    cu_global(in + arg0, arg1, angle, steps, dist);
}

__global__ void cuda_transpose_kernel(cpx *in, cpx *out, int n)
{
    // Banking issues when CU_TILE_DIM % WARP_SIZE == 0, current WARP_SIZE == 32
    __shared__ cpx tile[CU_TILE_DIM][CU_TILE_DIM + 1];

    // Image offset
    int offset = gridDim.x * CU_TILE_DIM;
    offset = (blockIdx.z * offset * offset);
    in += offset;
    out += offset;

    // Write to shared from Global (in)
    int x = blockIdx.x * CU_TILE_DIM + threadIdx.x;
    int y = blockIdx.y * CU_TILE_DIM + threadIdx.y;
#pragma unroll
    for (int j = 0; j < CU_TILE_DIM; j += CU_BLOCK_DIM)
        for (int i = 0; i < CU_TILE_DIM; i += CU_BLOCK_DIM)
            tile[threadIdx.y + j][threadIdx.x + i] = in[(y + j) * n + (x + i)];

    SYNC_THREADS;
    // Write to global
    x = blockIdx.y * CU_TILE_DIM + threadIdx.x;
    y = blockIdx.x * CU_TILE_DIM + threadIdx.y;
#pragma unroll
    for (int j = 0; j < CU_TILE_DIM; j += CU_BLOCK_DIM)
        for (int i = 0; i < CU_TILE_DIM; i += CU_BLOCK_DIM)
            out[(y + j) * n + (x + i)] = tile[threadIdx.x + i][threadIdx.y + j];
}
#endif