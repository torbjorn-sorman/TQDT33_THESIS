#include "hip/hip_runtime.h"
#include "cuda_helper.cuh"
#if defined(_NVIDIA)
void checkCudaError(char *msg)
{
    hipError_t e;
    if (e = hipGetLastError()) printf("%s:\n%s: %s\n", msg, hipGetErrorName(e), hipGetErrorString(e));
}

void checkCudaError()
{
    hipError_t e;
    if (e = hipGetLastError()) printf("%s: %s\n", hipGetErrorName(e), hipGetErrorString(e));
}

#define ERROR_MARGIN 0.0001

// Useful functions for debugging
void console_print(cpx *seq, int n)
{
    for (int i = 0; i < n; ++i) printf("%f\t%f\n", seq[i].x, seq[i].y);
}

void console_print_cpx_img(cpx *seq, int n)
{
    printf("\n");
    for (int y = 0; y < n; ++y) {
        for (int x = 0; x < n; ++x) {
            printf("%.2f\t", seq[y * n + x].x);
        }
        printf("\n");
    }
    printf("\n");
    for (int y = 0; y < n; ++y) {
        for (int x = 0; x < n; ++x) {
            printf("%.2f\t", seq[y * n + x].y);
        }
        printf("\n");
    }
}

void _cudaMalloc(int n, cpx **dev_in, cpx **dev_out)
{
    *dev_in = 0;    
    hipMalloc((void**)dev_in, n * sizeof(cpx));
    if (dev_out != NULL) {
        *dev_out = 0;
        hipMalloc((void**)dev_out, n * sizeof(cpx));
    }
}

void cuda_setup_buffers(int n, cpx **dev_in, cpx **dev_out, cpx **in, cpx **ref, cpx **out)
{
    _cudaMalloc(n, dev_in, dev_out);
    if (in == NULL && ref == NULL && out == NULL)
        return;
    fft_alloc_sequences(n, in, ref, out);
}

void _cudaFree(cpx **dev_in, cpx **dev_out)
{
    hipFree(*dev_in);
    if (dev_out != NULL)
        hipFree(*dev_out);
}

int cuda_shakedown(int n, cpx **dev_in, cpx **dev_out, cpx **in, cpx **ref, cpx **out)
{
    _cudaFree(dev_in, dev_out);
    hipDeviceSynchronize();
    if (in == NULL && ref == NULL && out == NULL)
        return 0;
    double diff = diff_seq(*in, *ref, n);
    free_all(*in, *out, *ref);
    hipError_t cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }
    return diff > ERROR_MARGIN;
}

void cuda_setup_buffers_2d(cpx **in, cpx **ref, cpx **dev_i, cpx **dev_o, size_t *size, int n)
{
    char input_file[40];
    sprintf_s(input_file, 40, "Images/%u.ppm", n);
    int sz;
    *in = (cpx *)malloc(sizeof(cpx) * n * n); 
    read_image(*in, input_file, &sz);
    *ref = (cpx *)malloc(sizeof(cpx) * n * n);
    memcpy(*ref, *in, sizeof(cpx) * n * n);
    *size = n * n * sizeof(cpx);
    hipMalloc((void**)dev_i, *size);
    hipMalloc((void**)dev_o, *size);
    hipMemcpy(*dev_i, *in, *size, hipMemcpyHostToDevice);
}

void cuda_shakedown_2d(cpx **in, cpx **ref, cpx **dev_i, cpx **dev_o)
{    
    free_all(*in, *ref);
    hipFree(*dev_i);
    hipFree(*dev_o);
    hipError_t cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
    }
}

int cuda_compare_result(cpx *in, cpx *ref, cpx *dev, size_t size, int len)
{
    hipMemcpy(in, dev, size, hipMemcpyDeviceToHost);
    for (int i = 0; i < len; ++i) {
        if (hipCabsf(hipCsubf(in[i], ref[i])) > 0.0001) {
            return 0;
        }
    }
    return 1;
}

#define max(a, b) ((a) > (b) ? (a) : (b))

int cuda_compare_result(cpx *in, cpx *ref, cpx *dev, size_t size, int len, double *relDiff)
{
    double mDiff = 0.0;
    double mVal = -1;
    hipMemcpy(in, dev, size, hipMemcpyDeviceToHost);
    for (int i = 0; i < len; ++i) {
        mVal = max(mVal, max(hipCabsf(in[i]), hipCabsf(ref[i])));
        double tmp = hipCabsf(hipCsubf(in[i], ref[i]));
        mDiff = tmp > mDiff ? tmp : mDiff;
    }
    *relDiff = (mDiff / mVal);
    return *relDiff < 0.00001;
}

void cudaCheckError(hipError_t err)
{
    if (err) {
        printf("\n%s: %s\n", hipGetErrorName(err), hipGetErrorString(err));
        getchar();
        exit(err);
    }
}

void cudaCheckError()
{
    cudaCheckError(hipGetLastError());
}
#endif