#include "hip/hip_runtime.h"
#include "cuda_helper.cuh"
#if defined(_NVIDIA)
void checkCudaError(char *msg)
{
    hipError_t e;
    if (e = hipGetLastError()) printf("%s:\n%s: %s\n", msg, hipGetErrorName(e), hipGetErrorString(e));
}

void checkCudaError()
{
    hipError_t e;
    if (e = hipGetLastError()) printf("%s: %s\n", hipGetErrorName(e), hipGetErrorString(e));
}

#define ERROR_MARGIN 0.0001

// Useful functions for debugging
void console_print(cpx *seq, int n)
{
    for (int i = 0; i < n; ++i) printf("%f\t%f\n", seq[i].x, seq[i].y);
}

void console_print_cpx_img(cpx *seq, int n)
{
    printf("\n");
    for (int y = 0; y < n; ++y) {
        for (int x = 0; x < n; ++x) {
            printf("%.2f\t", seq[y * n + x].x);
        }
        printf("\n");
    }
    printf("\n");
    for (int y = 0; y < n; ++y) {
        for (int x = 0; x < n; ++x) {
            printf("%.2f\t", seq[y * n + x].y);
        }
        printf("\n");
    }
}

void cuda_setup_buffers(int n, cpx **dev_in, cpx **dev_out, cpx **in, cpx **ref, cpx **out)
{
    size_t total_size = sizeof(cpx) * batch_size(n);
    if (dev_in)  { *dev_in = 0;  hipMalloc((void**)dev_in,  total_size); }
    if (dev_out) { *dev_out = 0; hipMalloc((void**)dev_out, total_size); }    
    setup_seq(in, out, ref, batch_count(n), n);
}

void _cudaFree(cpx **dev_in, cpx **dev_out)
{
    hipFree(*dev_in);
    if (dev_out != NULL)
        hipFree(*dev_out);
}

int cuda_shakedown(int n, cpx **dev_in, cpx **dev_out, cpx **in, cpx **ref, cpx **out)
{
    _cudaFree(dev_in, dev_out);
    hipDeviceSynchronize();
    double diff;
    if (in && ref) {
        diff = diff_seq(*in, *ref, batch_size(n));
        free_all(*in, *ref);
    }
    if (out) {
        free(*out);
    }
    hipError_t cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }
    return diff > ERROR_MARGIN;
}

void cuda_setup_buffers_2d(cpx **in, cpx **ref, cpx **dev_i, cpx **dev_o, size_t *size, int n)
{
    setup_seq_2d(in, NULL, ref, batch_count(n), n);
    *size = batch_size(n * n) * sizeof(cpx);
    hipMalloc((void**)dev_i, *size);
    hipMalloc((void**)dev_o, *size);
    hipMemcpy(*dev_i, *in, *size, hipMemcpyHostToDevice);
}

void cuda_shakedown_2d(cpx **in, cpx **ref, cpx **dev_i, cpx **dev_o)
{    
    free_all(*in, *ref);
    hipFree(*dev_i);
    hipFree(*dev_o);
    hipError_t cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
    }
}

int cuda_compare_result(cpx *in, cpx *ref, cpx *dev, size_t size, int len)
{
    hipMemcpy(in, dev, size, hipMemcpyDeviceToHost);
    for (int i = 0; i < len; ++i) {
        if (hipCabsf(hipCsubf(in[i], ref[i])) > 0.0001) {
            return 0;
        }
    }
    return 1;
}

#define max(a, b) ((a) > (b) ? (a) : (b))

int cuda_compare_result(cpx *in, cpx *ref, cpx *dev, size_t size, int len, double *relDiff)
{
    double mDiff = 0.0;
    double mVal = -1;
    hipMemcpy(in, dev, size, hipMemcpyDeviceToHost);
    for (int i = 0; i < len; ++i) {
        mVal = max(mVal, max(hipCabsf(in[i]), hipCabsf(ref[i])));
        double tmp = hipCabsf(hipCsubf(in[i], ref[i]));
        mDiff = tmp > mDiff ? tmp : mDiff;
    }
    *relDiff = (mDiff / mVal);
    return *relDiff < 0.00001;
}

void cudaCheckError(hipError_t err)
{
    if (err) {
        printf("\n%s: %s\n", hipGetErrorName(err), hipGetErrorString(err));
        getchar();
        exit(err);
    }
}

void cudaCheckError()
{
    cudaCheckError(hipGetLastError());
}
#endif