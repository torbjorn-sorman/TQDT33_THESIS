#include "hip/hip_runtime.h"
#include "cuda_helper.cuh"

__global__ void kernelTranspose(cpx *in, cpx *out, int n)
{
    // Banking issues when TILE_DIM % WARP_SIZE == 0, current WARP_SIZE == 32
    __shared__ cpx tile[TILE_DIM][TILE_DIM + 1];

    // Write to shared from Global (in)
    int x = blockIdx.x * TILE_DIM + threadIdx.x;
    int y = blockIdx.y * TILE_DIM + threadIdx.y;
    for (int j = 0; j < TILE_DIM; j += THREAD_TILE_DIM)
        for (int i = 0; i < TILE_DIM; i += THREAD_TILE_DIM)
            tile[threadIdx.y + j][threadIdx.x + i] = in[(y + j) * n + (x + i)];

    SYNC_THREADS;
    // Write to global
    x = blockIdx.y * TILE_DIM + threadIdx.x;
    y = blockIdx.x * TILE_DIM + threadIdx.y;
    for (int j = 0; j < TILE_DIM; j += THREAD_TILE_DIM)
        for (int i = 0; i < TILE_DIM; i += THREAD_TILE_DIM)
            out[(y + j) * n + (x + i)] = tile[threadIdx.x + i][threadIdx.y + j];
}

__global__ void kernelTranspose(cuSurf in, cuSurf out, int n)
{
    // Banking issues when TILE_DIM % WARP_SIZE == 0, current WARP_SIZE == 32
    __shared__ cpx tile[TILE_DIM][TILE_DIM + 1];

    // Write to shared from Global (in)
    int x = blockIdx.x * TILE_DIM + threadIdx.x;
    int y = blockIdx.y * TILE_DIM + threadIdx.y;
    for (int j = 0; j < TILE_DIM; j += THREAD_TILE_DIM)
        for (int i = 0; i < TILE_DIM; i += THREAD_TILE_DIM)
            SURF2D_READ(&(tile[threadIdx.y + j][threadIdx.x + i]), in, x + i, y + j);
    //tile[threadIdx.y + j][threadIdx.x + i] = in[(y + j) * n + (x + i)];

    SYNC_THREADS;
    // Write to global
    x = blockIdx.y * TILE_DIM + threadIdx.x;
    y = blockIdx.x * TILE_DIM + threadIdx.y;
    for (int j = 0; j < TILE_DIM; j += THREAD_TILE_DIM)
        for (int i = 0; i < TILE_DIM; i += THREAD_TILE_DIM)
            SURF2D_WRITE(tile[threadIdx.x + i][threadIdx.y + j], out, x + i, y + j);
    //out[(y + j) * n + (x + i)] = tile[threadIdx.x + i][threadIdx.y + j];
}

void set_block_and_threads(int *number_of_blocks, int *threadsPerBlock, int size)
{
    if (size > MAX_BLOCK_SIZE) {
        *number_of_blocks = size / MAX_BLOCK_SIZE;
        *threadsPerBlock = MAX_BLOCK_SIZE;
    }
    else {
        *number_of_blocks = 1;
        *threadsPerBlock = size;
    }
}

void set_block_and_threads2D(dim3 *number_of_blocks, int *threadsPerBlock, int n)
{
    number_of_blocks->x = n;
    int n_half = n >> 1;
    if (n_half > MAX_BLOCK_SIZE) {
        number_of_blocks->y = n_half / MAX_BLOCK_SIZE;
        *threadsPerBlock = MAX_BLOCK_SIZE;
    }
    else {
        number_of_blocks->y = 1;
        *threadsPerBlock = n_half;
    }
}

void set_block_and_threads_transpose(dim3 *bTrans, dim3 *tTrans, int n)
{
    int minDim = n > TILE_DIM ? (n / TILE_DIM) : 1;
    bTrans->z = tTrans->z = 1;
    bTrans->x = bTrans->y = minDim;
    tTrans->x = tTrans->y = THREAD_TILE_DIM;
}

void checkCudaError(char *msg)
{
    hipError_t e;
    if (e = hipGetLastError()) printf("%s:\n%s: %s\n", msg, hipGetErrorName(e), hipGetErrorString(e));
}

void checkCudaError()
{
    hipError_t e;
    if (e = hipGetLastError()) printf("%s: %s\n", hipGetErrorName(e), hipGetErrorString(e));
}

#define ERROR_MARGIN 0.0001

// Useful functions for debugging
void console_print(cpx *seq, int n)
{
    for (int i = 0; i < n; ++i) printf("%f\t%f\n", seq[i].x, seq[i].y);
}

void console_print_cpx_img(cpx *seq, int n)
{
    printf("\n");
    for (int y = 0; y < n; ++y) {
        for (int x = 0; x < n; ++x) {
            printf("%.2f\t", seq[y * n + x].x);
        }
        printf("\n");
    }
    printf("\n");
    for (int y = 0; y < n; ++y) {
        for (int x = 0; x < n; ++x) {
            printf("%.2f\t", seq[y * n + x].y);
        }
        printf("\n");
    }
}

cpx *get_sin_img(int n)
{
    cpx *seq;
    seq = (cpx *)malloc(sizeof(cpx) * n * n);
    for (int y = 0; y < n; ++y)
        for (int x = 0; x < n; ++x)
            seq[y * n + x] = make_hipFloatComplex((float)sin(M_2_PI * (((double)x) / n)), 0.f);
    return seq;
}

void _cudaMalloc(int n, cpx **dev_in, cpx **dev_out)
{
    *dev_in = 0;    
    hipMalloc((void**)dev_in, n * sizeof(cpx));
    if (dev_out != NULL) {
        *dev_out = 0;
        hipMalloc((void**)dev_out, n * sizeof(cpx));
    }
}

void _fftTestSeq(int n, cpx **in, cpx **ref, cpx **out)
{
    *in = get_seq(n, 1);
    *ref = get_seq(n, *in);
    *out = get_seq(n);
}

void cuda_setup_buffers(int n, cpx **dev_in, cpx **dev_out, cpx **in, cpx **ref, cpx **out)
{
    _cudaMalloc(n, dev_in, dev_out);
    if (in == NULL && ref == NULL && out == NULL)
        return;
    _fftTestSeq(n, in, ref, out);
}

void _cudaFree(cpx **dev_in, cpx **dev_out)
{
    hipFree(*dev_in);
    if (dev_out != NULL) hipFree(*dev_out);
}

void _fftFreeSeq(cpx **in, cpx **ref, cpx **out)
{
    free(*in);
    free(*ref);
    free(*out);
}

int cuda_shakedown(int n, cpx **dev_in, cpx **dev_out, cpx **in, cpx **ref, cpx **out)
{
    _cudaFree(dev_in, dev_out);
    hipDeviceSynchronize();
    if (in == NULL && ref == NULL && out == NULL)
        return 0;
    double diff = diff_seq(*in, *ref, n);
    _fftFreeSeq(in, out, ref);
    hipError_t cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }
    return diff > ERROR_MARGIN;
}

void cuda_setup_buffers_2d(cpx **in, cpx **ref, cpx **dev_i, cpx **dev_o, size_t *size, int n)
{
    char input_file[40];
    sprintf_s(input_file, 40, "Images/%u.ppm", n);
    int sz;
    *in = (cpx *)malloc(sizeof(cpx) * n * n); 
    read_image(*in, input_file, &sz);
    *ref = (cpx *)malloc(sizeof(cpx) * n * n);
    memcpy(*ref, *in, sizeof(cpx) * n * n);
    *size = n * n * sizeof(cpx);
    hipMalloc((void**)dev_i, *size);
    if (dev_o != NULL)
        hipMalloc((void**)dev_o, *size);
    hipMemcpy(*dev_i, *in, *size, hipMemcpyHostToDevice);
}

void cuda_shakedown_2d(cpx **in, cpx **ref, cpx **dev_i, cpx **dev_o)
{    
    free(*in);
    free(*ref);
    hipFree(*dev_i);
    if (dev_o != NULL)
        hipFree(*dev_o);
    hipError_t cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
    }
}

int cuda_compare_result(cpx *in, cpx *ref, cpx *dev, size_t size, int len)
{
    hipMemcpy(in, dev, size, hipMemcpyDeviceToHost);
    for (int i = 0; i < len; ++i) {
        if (hipCabsf(hipCsubf(in[i], ref[i])) > 0.0001) {
            return 0;
        }
    }
    return 1;
}

#define max(a, b) ((a) > (b) ? (a) : (b))

int cuda_compare_result(cpx *in, cpx *ref, cpx *dev, size_t size, int len, double *relDiff)
{
    double mDiff = 0.0;
    double mVal = -1;
    hipMemcpy(in, dev, size, hipMemcpyDeviceToHost);
    for (int i = 0; i < len; ++i) {
        mVal = max(mVal, max(hipCabsf(in[i]), hipCabsf(ref[i])));
        double tmp = hipCabsf(hipCsubf(in[i], ref[i]));
        mDiff = tmp > mDiff ? tmp : mDiff;
    }
    *relDiff = (mDiff / mVal);
    return *relDiff < 0.00001;
}

void cudaCheckError(hipError_t err)
{
    if (err) {
        printf("\n%s: %s\n", hipGetErrorName(err), hipGetErrorString(err));
        getchar();
        exit(err);
    }
}

void cudaCheckError()
{
    cudaCheckError(hipGetLastError());
}

void fft2DSurfSetup(cpx **in, cpx **ref, size_t *size, int sinus, int n, hipArray **cuInputArray, hipArray **cuOutputArray, cuSurf *inputSurfObj, cuSurf *outputSurfObj)
{
    if (sinus) {
        *in = get_sin_img(n);
        *ref = get_sin_img(n);
    }
    else {
        char input_file[40];
        sprintf_s(input_file, 40, "Images/%u.ppm", n);
        int sz;
        *in = (cpx *)malloc(sizeof(cpx) * n * n);
        read_image(*in, input_file, &sz);
        *ref = (cpx *)malloc(sizeof(cpx) * n * n);
        memcpy(*ref, *in, sizeof(cpx) * n * n);
    }
    *size = n * n * sizeof(cpx);
    // Allocate CUDA arrays in device memory
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float2>();
    hipMallocArray(cuInputArray, &channelDesc, n, n, hipArraySurfaceLoadStore);
    cudaCheckError();
    if (cuOutputArray != NULL) {
        hipMallocArray(cuOutputArray, &channelDesc, n, n, hipArraySurfaceLoadStore);
    }
    // Specify surface
    struct hipResourceDesc resDesc;
    memset(&resDesc, 0, sizeof(resDesc));
    resDesc.resType = hipResourceTypeArray;
    // Create the surface objects
    resDesc.res.array.array = *cuInputArray;
    *inputSurfObj = 0;
    hipCreateSurfaceObject(inputSurfObj, &resDesc);
    cudaCheckError();
    if (outputSurfObj != NULL) {
        resDesc.res.array.array = *cuOutputArray;
        *outputSurfObj = 0;
        hipCreateSurfaceObject(outputSurfObj, &resDesc);
        cudaCheckError();
    }
}