#include "hip/hip_runtime.h"
#include <stdio.h>
#include <>

#include "math.h"
#include "FFTConstantGeom.cuh"
#include "fft_helper.cuh"

__global__ void _FFTBody(hipFloatComplex *in, hipFloatComplex *out, hipFloatComplex *W, unsigned int mask, const int n2);
__global__ void _FFTBody(hipFloatComplex *in, hipFloatComplex *out, const float w_angle, unsigned int mask, const int n2);
__host__ __inline void _setBlocksAndThreads(int *numBlocks, int *threadsPerBlock, const int size);
__host__ __inline void _swap(hipFloatComplex **in, hipFloatComplex **out);

__host__ void FFTConstGeom(fftDirection dir, hipFloatComplex *dev_in, hipFloatComplex *dev_out, hipFloatComplex *dev_W, unsigned int *buf, const int n)
{
    int steps, depth, threadsPerBlock, numBlocks;
    const float w_angle = dir * (M_2_PI / n);
    const float scale = dir == FFT_FORWARD ? 1.f : 1.f / n;
    const int n2 = (n / 2);
    hipFloatComplex *W;

    depth = log2_32(n);

    _setBlocksAndThreads(&numBlocks, &threadsPerBlock, n2);
    twiddle_factors << < numBlocks, threadsPerBlock >> >(dev_W, w_angle, n);
    hipDeviceSynchronize();

    steps = 0;
    _FFTBody << < numBlocks, threadsPerBlock >> >(dev_in, dev_out, dev_W, 0xffffffff << steps, n2);
    hipDeviceSynchronize();
    while (++steps < depth) {
        _swap(&dev_in, &dev_out);
        _FFTBody << < numBlocks, threadsPerBlock >> >(dev_in, dev_out, dev_W, 0xffffffff << steps, n2);
        hipDeviceSynchronize();
    }

    _setBlocksAndThreads(&numBlocks, &threadsPerBlock, n);
    bit_reverse << < numBlocks, threadsPerBlock >> >(dev_out, dev_in, scale, 32 - depth);
    hipDeviceSynchronize();

    *buf = (depth - 1) % 2;
}

__host__ void FFTConstGeom2(fftDirection dir, hipFloatComplex *dev_in, hipFloatComplex *dev_out, unsigned int *buf, const int n)
{
    int steps, depth, threadsPerBlock, numBlocks;
    const float w_angle = dir * (M_2_PI / n);
    const float scale = dir == FFT_FORWARD ? 1.f : 1.f / n;
    const int n2 = (n / 2);

    depth = log2_32(n);
    steps = 0;

    _setBlocksAndThreads(&numBlocks, &threadsPerBlock, n2);
    _FFTBody << < numBlocks, threadsPerBlock >> >(dev_in, dev_out, w_angle, 0xffffffff << steps, n2);
    hipDeviceSynchronize();
    while (++steps < depth) {
        _swap(&dev_in, &dev_out);
        _FFTBody << < numBlocks, threadsPerBlock >> >(dev_in, dev_out, w_angle, 0xffffffff << steps, n2);
        hipDeviceSynchronize();
    }

    _setBlocksAndThreads(&numBlocks, &threadsPerBlock, n);
    bit_reverse << < numBlocks, threadsPerBlock >> >(dev_out, dev_in, scale, 32 - depth);
    hipDeviceSynchronize();

    *buf = (depth - 1) % 2;
}

__global__ void _FFTBody(hipFloatComplex *in, hipFloatComplex *out, hipFloatComplex *W, unsigned int mask, const int n2)
{
    int threadId = (blockIdx.x * blockDim.x + threadIdx.x);
    int i = threadId * 2;
    int u = n2 + threadId;
    hipFloatComplex tmp = hipCsubf(in[threadId], in[u]);
    out[i] = hipCaddf(in[threadId], in[u]);
    out[i + 1] = hipCmulf(W[threadId & mask], tmp);
}

__global__ void _FFTBody(hipFloatComplex *in, hipFloatComplex *out, const float w_angle, unsigned int mask, const int n2)
{
    int threadId = (blockIdx.x * blockDim.x + threadIdx.x);
    int i = threadId * 2;
    int u = n2 + threadId;
    hipFloatComplex tmp = hipCsubf(in[threadId], in[u]);
    hipFloatComplex twiddle;
    sincosf(w_angle * (threadId & mask), &twiddle.y, &twiddle.x);
    out[i] = hipCaddf(in[threadId], in[u]);
    out[i + 1] = hipCmulf(twiddle, tmp);
}

__host__ __inline void _setBlocksAndThreads(int *numBlocks, int *threadsPerBlock, const int size)
{
    int v1 = 256;
    if (size > v1) {
        *numBlocks = size / v1;
        *threadsPerBlock = v1;
    }
    else {
        *numBlocks = 1;
        *threadsPerBlock = size;
    }
}

__host__ __inline void _swap(hipFloatComplex **in, hipFloatComplex **out)
{
    hipFloatComplex *tmp = *in;
    *in = *out;
    *out = tmp;
}