#include <stdio.h>
#include <hip/hip_runtime.h>
#include <>
#include <math.h>

#include <hipfft/hipfft.h>

#include "definitions.cuh"
#include "fft_test.cuh"
#include "fft_helper.cuh"

#include "FFTConstGeom.cuh"
#include "FFTRegular.cuh"
#include "FFTTobb.cuh"

__host__ double cuFFT_Performance(const size_t n);

int main()
{    
    printf("\tcuFFT\ttbFFT\ttbFFT\n");
    for (int n = power2(2); n < power2(16); n *= 2) {        
        printf("\n%d:", n);

        // cuFFT
        printf("\t%.0f", cuFFT_Performance(n));

        // Regular (not working, used as ref)
        printf("\t%.0f", FFTRegular_Performance(n));
        if (FFTRegular_Validate(n) == 0) printf("!");        

        // Regular
        printf("\t%.0f", FFTTobb_Performance(n));
        if (FFTTobb_Validate(n) == 0) printf("!");

        // Const geom
        printf("\t%.0f", FFTConstGeom_Performance(n));
        if (FFTConstGeom_Validate(n) == 0) printf("!");
    }
    printf("\nDone...");
    getchar();
    return 0;
}

__host__ double cuFFT_Performance(const size_t n)
{
    double measures[NUM_PERFORMANCE];
    cpx *dev_in = 0;
    cpx *dev_out = 0;
    hipMalloc((void**)&dev_in, sizeof(cpx) * n);
    hipMalloc((void**)&dev_out, sizeof(cpx) * n);
    hipfftHandle plan;
    hipfftPlan1d(&plan, n, HIPFFT_C2C, 1);
    for (int i = 0; i < 20; ++i) {
        startTimer();
        hipfftExecC2C(plan, dev_in, dev_out, HIPFFT_FORWARD);
        hipDeviceSynchronize();
        measures[i] = stopTimer();
    }
    hipfftDestroy(plan);
    hipFree(dev_in);
    hipFree(dev_out);
    return avg(measures, NUM_PERFORMANCE);
}

__host__ double cuFFT_2D_Performance(const size_t n)
{
    double measures[NUM_PERFORMANCE];
    cpx *dev_in = 0;
    cpx *dev_out = 0;
    hipMalloc((void**)&dev_in, sizeof(cpx) * n * n);
    hipMalloc((void**)&dev_out, sizeof(cpx) * n * n);
    hipfftHandle plan;
    hipfftPlan2d(&plan, n, n, HIPFFT_C2C);
    for (int i = 0; i < 20; ++i) {
        startTimer();
        hipfftExecC2C(plan, dev_in, dev_out, HIPFFT_FORWARD);
        hipDeviceSynchronize();
        measures[i] = stopTimer();
    }
    hipfftDestroy(plan);
    hipFree(dev_in);
    hipFree(dev_out);
    return avg(measures, NUM_PERFORMANCE);
}