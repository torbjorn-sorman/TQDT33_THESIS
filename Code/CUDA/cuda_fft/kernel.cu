
//#define PROFILER
//#define IMAGE_TEST

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <>
#include <math.h>

#ifndef PROFILER
#include <hipfft/hipfft.h>
#endif

#include "tsDefinitions.cuh"
#include "tsTest.cuh"

#include "tsCombine.cuh"
#include "tsCombineGPUSync.cuh"
#include "tsCombineGPUSyncTex.cuh"

#ifndef PROFILER

__host__ double cuFFT_Performance(int n);
__host__ double cuFFT_2D_Performance(int n);
__host__ void toFile(const char *name, const double m[], int ms);

// Print device properties
void printDevProp(hipDeviceProp_t devProp)
{
    printf("Major revision number:         %d\n", devProp.major);
    printf("Minor revision number:         %d\n", devProp.minor);
    printf("Name:                          %s\n", devProp.name);
    printf("Total global memory:           %u\n", devProp.totalGlobalMem);
    printf("Total shared memory per block: %u\n", devProp.sharedMemPerBlock);
    printf("Total registers per block:     %d\n", devProp.regsPerBlock);
    printf("Warp size:                     %d\n", devProp.warpSize);
    printf("Maximum memory pitch:          %u\n", devProp.memPitch);
    printf("Maximum threads per block:     %d\n", devProp.maxThreadsPerBlock);
    for (int i = 0; i < 3; ++i)
        printf("Maximum dimension %d of block:  %d\n", i, devProp.maxThreadsDim[i]);
    for (int i = 0; i < 3; ++i)
        printf("Maximum dimension %d of grid:   %d\n", i, devProp.maxGridSize[i]);
    printf("Clock rate:                    %d\n", devProp.clockRate);
    printf("Total constant memory:         %u\n", devProp.totalConstMem);
    printf("Texture alignment:             %u\n", devProp.textureAlignment);
    printf("Concurrent copy and execution: %s\n", (devProp.deviceOverlap ? "Yes" : "No"));
    printf("Number of multiprocessors:     %d\n", devProp.multiProcessorCount);
    printf("Kernel execution timeout:      %s\n", (devProp.kernelExecTimeoutEnabled ? "Yes" : "No"));
    return;
}

#endif

#define RUNS 16

int main()
{    
    //hipDeviceProp_t prop;
    //hipGetDeviceProperties(&prop, 0);
    //printDevProp(prop);
    hipDeviceSetCacheConfig(hipFuncCachePreferShared);
    hipDeviceSetSharedMemConfig(hipSharedMemBankSizeEightByte);
        
#if defined(PROFILER)
    int start = 2;
    int end = start + RUNS;
    for (unsigned int n = power2(start); n < power2(end); n *= 2)
        tsCombine_Performance(n);
#elif defined(IMAGE_TEST)
    printf("\n2D validation & performance test!\n");
    printf("\tn\tcuFFT\tMy\tMy Surf\t");
    for (unsigned int n = TILE_DIM / 2; n <= 4096; n *= 2) {
        printf("\n\t%d:", n);
        printf("\t%.0f", cuFFT_2D_Performance(n));

        if (n < 4096) {            
            tsCombineGPUSync2D_Test(n);
            tsCombineGPUSyncTex2D_Test(n);
        }
    }
    getchar();
#else
    int start = 2;
    int end = start + RUNS;
    int index = 0;
    double cuFFTm[RUNS];
    double combineFFTm[RUNS];
    printf("\n\t\tcuFFT\tComb");
    printf("\n");
    for (unsigned int n = power2(start); n < power2(end); n *= 2) {        
        printf("\n%d:", n);
        
        char *fmt = n > 1000000 ? "\t%.0f" : "\t\t%.0f";
        
        // cuFFT
        printf(fmt, cuFFTm[index] = cuFFT_Performance(n));
                
        // Combine
        printf("\t%.0f", combineFFTm[index] = tsCombine_Performance(n));
        if (tsCombine_Validate(n) == 0) printf("!");
        
        ++index;
    }
    printf("\n\n");
    toFile("cuFFT", cuFFTm, RUNS);
    toFile("Block Combine CPU Sync", combineFFTm, RUNS);
    
    printf("\nDone...");
    getchar();
#endif
    return 0;
}

#ifndef PROFILER

__host__ double cuFFT_Performance(int n)
{
    double measures[NUM_PERFORMANCE];
    cpx *dev_in,*dev_out;
    fftMalloc(n, &dev_in, &dev_out, NULL, NULL, NULL, NULL);

    hipfftHandle plan;
    hipfftPlan1d(&plan, n, HIPFFT_C2C, 1);
    for (int i = 0; i < 20; ++i) {
        startTimer();
        hipfftExecC2C(plan, dev_in, dev_out, HIPFFT_FORWARD);
        hipDeviceSynchronize();
        measures[i] = stopTimer();
    }
    hipfftDestroy(plan);
    fftResultAndFree(n, &dev_in, &dev_out, NULL, NULL, NULL, NULL);
    return avg(measures, NUM_PERFORMANCE);
}

__host__ double cuFFT_2D_Performance(int n)
{
    double measures[NUM_PERFORMANCE];
    cpx *dev_in, *dev_out;
    fftMalloc(n * n, &dev_in, &dev_out, NULL, NULL, NULL, NULL);
    hipfftHandle plan;
    hipfftPlan2d(&plan, n, n, HIPFFT_C2C);
    for (int i = 0; i < 20; ++i) {
        startTimer();
        hipfftExecC2C(plan, dev_in, dev_out, HIPFFT_FORWARD);
        hipDeviceSynchronize();
        measures[i] = stopTimer();
    }
    hipfftDestroy(plan);
    fftResultAndFree(n, &dev_in, &dev_out, NULL, NULL, NULL, NULL);
    return avg(measures, NUM_PERFORMANCE);
}

__host__ void toFile(const char *name, const double m[], int ms)
{
    char filename[64] = "";
    FILE *f;
    strcat_s(filename, "out/");
    strcat_s(filename, name);
    strcat_s(filename, ".txt");
    fopen_s(&f, filename, "w");
    for (int i = 0; i < ms; ++i)
        fprintf_s(f, "%0.f\n", m[i]);

    printf("File '%s' written.\n", filename);
    fclose(f);
}

#endif