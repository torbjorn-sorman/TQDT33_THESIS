
//#define PROFILER

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <>
#include <math.h>

#ifndef PROFILER
#include <hipfft/hipfft.h>
#endif

#include "tsDefinitions.cuh"
#include "tsTest.cuh"

#include "tsConstantGeometry.cuh"
#include "tsConstantGeometry_SB.cuh"
#include "tsTobb.cuh"
#include "tsTobb_SB.cuh"
#include "tsCombine.cuh"

#ifndef PROFILER

__host__ double cuFFT_Performance(const int n);
void toFile(const char *name, const double m[], const int ms);

// Print device properties
void printDevProp(hipDeviceProp_t devProp)
{
    printf("Major revision number:         %d\n", devProp.major);
    printf("Minor revision number:         %d\n", devProp.minor);
    printf("Name:                          %s\n", devProp.name);
    printf("Total global memory:           %u\n", devProp.totalGlobalMem);
    printf("Total shared memory per block: %u\n", devProp.sharedMemPerBlock);
    printf("Total registers per block:     %d\n", devProp.regsPerBlock);
    printf("Warp size:                     %d\n", devProp.warpSize);
    printf("Maximum memory pitch:          %u\n", devProp.memPitch);
    printf("Maximum threads per block:     %d\n", devProp.maxThreadsPerBlock);
    for (int i = 0; i < 3; ++i)
        printf("Maximum dimension %d of block:  %d\n", i, devProp.maxThreadsDim[i]);
    for (int i = 0; i < 3; ++i)
        printf("Maximum dimension %d of grid:   %d\n", i, devProp.maxGridSize[i]);
    printf("Clock rate:                    %d\n", devProp.clockRate);
    printf("Total constant memory:         %u\n", devProp.totalConstMem);
    printf("Texture alignment:             %u\n", devProp.textureAlignment);
    printf("Concurrent copy and execution: %s\n", (devProp.deviceOverlap ? "Yes" : "No"));
    printf("Number of multiprocessors:     %d\n", devProp.multiProcessorCount);
    printf("Kernel execution timeout:      %s\n", (devProp.kernelExecTimeoutEnabled ? "Yes" : "No"));
    return;
}

#endif

#define RUNS 20

int main()
{    
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    hipDeviceSetCacheConfig(hipFuncCachePreferShared);
    hipDeviceSetSharedMemConfig(hipSharedMemBankSizeEightByte);
    //printDevProp(prop);
    
    int start = 2;
    int end = start + RUNS;
    int index = 0;
    double cuFFTm[RUNS];
    double constgeomFFTm[RUNS];
    double combineFFTm[RUNS];

#ifdef PROFILER
    for (unsigned int n = power2(start); n < power2(end); n *= 2)
        tsCombine_Performance(n);
#else
    printf("\n\t\tcuFFT\tConst\tComb\tTobbSB\tConstSB\n");    
    for (unsigned int n = power2(start); n < power2(end); n *= 2) {        
        printf("\n%d:", n);
        
        char *fmt = n > 1000000 ? "\t%.0f" : "\t\t%.0f";

        // cuFFT
        printf(fmt, cuFFTm[index] = cuFFT_Performance(n));
                
        // Const geom
        printf("\t%.0f", constgeomFFTm[index] = tsConstantGeometry_Performance(n));
        if (tsConstantGeometry_Validate(n) == 0) printf("!");

        // Combine
        printf("\t%.0f", combineFFTm[index] = tsCombine_Performance(n));
        if (tsCombine_Validate(n) == 0) printf("!");

        if (n <= MAX_BLOCK_SIZE * 2) {        
            // Tobb
            printf("\t%.0f", tsTobb_SB_Performance(n));
            if (tsTobb_SB_Validate(n) == 0) printf("!");
            // Const geom
            printf("\t%.0f", tsConstantGeometry_SB_Performance(n));
            if (tsConstantGeometry_SB_Validate(n) == 0) printf("!");
        }

        ++index;
    }
    printf("\n\n");
    toFile("cuFFT", cuFFTm, RUNS);
    toFile("constant geometry", constgeomFFTm, RUNS);
    toFile("block combine", combineFFTm, RUNS);
    printf("\nDone...");
    getchar();
#endif
    return 0;
}

#ifndef PROFILER

__host__ double cuFFT_Performance(const int n)
{
    double measures[NUM_PERFORMANCE];
    cpx *dev_in,*dev_out;
    fftMalloc(n, &dev_in, &dev_out, NULL, NULL, NULL, NULL);

    hipfftHandle plan;
    hipfftPlan1d(&plan, n, HIPFFT_C2C, 1);
    for (int i = 0; i < 20; ++i) {
        startTimer();
        hipfftExecC2C(plan, dev_in, dev_out, HIPFFT_FORWARD);
        hipDeviceSynchronize();
        measures[i] = stopTimer();
    }
    hipfftDestroy(plan);
    fftResultAndFree(n, &dev_in, &dev_out, NULL, NULL, NULL, NULL);
    return avg(measures, NUM_PERFORMANCE);
}

__host__ double cuFFT_2D_Performance(const int n)
{
    double measures[NUM_PERFORMANCE];
    cpx *dev_in, *dev_out;
    fftMalloc(n, &dev_in, &dev_out, NULL, NULL, NULL, NULL);
    hipfftHandle plan;
    hipfftPlan2d(&plan, n, n, HIPFFT_C2C);
    for (int i = 0; i < 20; ++i) {
        startTimer();
        hipfftExecC2C(plan, dev_in, dev_out, HIPFFT_FORWARD);
        hipDeviceSynchronize();
        measures[i] = stopTimer();
    }
    hipfftDestroy(plan);
    fftResultAndFree(n, &dev_in, &dev_out, NULL, NULL, NULL, NULL);
    return avg(measures, NUM_PERFORMANCE);
}

void toFile(const char *name, const double m[], const int ms)
{
    char filename[64] = "";
    FILE *f;
    strcat_s(filename, "out/");
    strcat_s(filename, name);
    strcat_s(filename, ".txt");
    fopen_s(&f, filename, "w");
    for (int i = 0; i < ms; ++i)
        fprintf_s(f, "%0.f\n", m[i]);

    printf("File '%s' written.\n", filename);
    fclose(f);
}

#endif