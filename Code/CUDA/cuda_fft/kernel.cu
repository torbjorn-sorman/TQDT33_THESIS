#include "hip/hip_runtime.h"
#include <stdio.h>
#include <Windows.h>
#include <hip/hip_runtime.h>
#include <>
#include <math.h>

#include <hipfft/hipfft.h>

#include "definitions.cuh"
#include "fft_test.cuh"
#include "fft_helper.cuh"

#include "FFTConstantGeom.cuh"
#include "FFTRegular.cuh"
#include "FFTTobb.cuh"

#define NO_TESTS 32
#define MAX_LENGTH 2097152 / 2

/* Performance measure on Windows, result in micro seconds */

LARGE_INTEGER StartingTime, EndingTime, ElapsedMicroseconds, Frequency;
#define QPF QueryPerformanceFrequency
#define QPC QueryPerformanceCounter
#define START_TIME QPF(&Frequency); QPC(&StartingTime)
#define STOP_TIME(RES) QPC(&EndingTime); ElapsedMicroseconds.QuadPart = EndingTime.QuadPart - StartingTime.QuadPart; ElapsedMicroseconds.QuadPart *= 1000000; ElapsedMicroseconds.QuadPart /= Frequency.QuadPart;(RES) = (double)ElapsedMicroseconds.QuadPart

hipError_t FFT_CUDA(fftDirection direction, hipFloatComplex *in, hipFloatComplex *out, double measures[], int n);
hipError_t FFT_ConstantGeometry(fftDirection direction, hipFloatComplex *in, hipFloatComplex *out, hipFloatComplex *W, double measures[], int n);
hipError_t FFT_ConstantGeometry2(fftDirection direction, hipFloatComplex *in, hipFloatComplex *out, double measures[], int n);
hipError_t FFT_Tobb(fftDirection direction, hipFloatComplex *in, hipFloatComplex *out, double measures[], int n);

unsigned int power(unsigned int base, int exp)
{
    if (exp == 0)
        return 1;
    unsigned int value = base;
    for (int i = 0; i < exp; ++i) {
        value *= base;
    }
    return value;
}

unsigned int power2(int exp)
{
    return power(2, exp);
}

int main()
{
    int n;
    double measures[20];
    hipFloatComplex *in, *out, *ref_in, *ref_out, *W;
    hipError_t cudaStatus;
    
    printf("\tcuFFT\ttbFFT\ttbFFT\n");
    for (n = power2(2); n < power2(19); n *= 2) {
        in = get_seq(n, 1);
        ref_in = get_seq(n, in);
        out = get_seq(n);
        ref_out = get_seq(n);
        W = get_seq(n);
        
        // cuFFT
        cudaStatus = FFT_CUDA(FFT_FORWARD, ref_in, ref_out, measures, n);
        printf("%d:\t%.0f", n, avg(measures, 20));
        cudaStatus = FFT_CUDA(FFT_INVERSE, ref_out, ref_in, measures, n);
        
        cudaStatus = FFT_ConstantGeometry2(FFT_FORWARD, in, out, measures, n);
        printf("\t%.0f", avg(measures, 20));
        cudaStatus = FFT_ConstantGeometry2(FFT_INVERSE, out, in, measures, n);        
        if (checkError(in, ref_in, (float)n, n, 0) == 1) printf("!");

        free(in);
        in = get_seq(n, 1);
        cudaStatus = FFT_ConstantGeometry(FFT_FORWARD, in, out, W, measures, n);
        printf("\t%.0f", avg(measures, 20));
        cudaStatus = FFT_ConstantGeometry(FFT_INVERSE, out, in, W, measures, n);
        if (checkError(in, ref_in, (float)n, n, 0) == 1) printf("!");

        free(in);
        in = get_seq(n, 1);
        cudaStatus = FFT_Tobb(FFT_FORWARD, in, out, measures, n);
        printf("\t%.0f", avg(measures, 20));
        cudaStatus = FFT_Tobb(FFT_INVERSE, out, in, measures, n);
        if (checkError(in, ref_in, (float)n, n, 0) == 1) printf("!");
                
        printf("\n");
        free(in);
        free(out);
        free(W);
        free(ref_in);
        free(ref_out);
    }
    printf("\n");

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        getchar();
        return 1;
    }

    getchar();
    return 0;
}

hipError_t FFT_CUDA(fftDirection direction, hipFloatComplex *in, hipFloatComplex *out, double measures[], int n)
{
    hipfftHandle plan;
    hipFloatComplex *dev_in;
    hipFloatComplex *dev_out;
    hipMalloc((void**)&dev_in, sizeof(hipFloatComplex) * n);
    hipMalloc((void**)&dev_out, sizeof(hipFloatComplex) * n);
    hipMemcpy(dev_in, in, n * sizeof(hipFloatComplex), hipMemcpyHostToDevice);
    hipfftPlan1d(&plan, n, HIPFFT_C2C, 1);        

    hipfftExecC2C(plan, dev_in, dev_out, direction);
    hipDeviceSynchronize();    
    
    hipMemcpy(out, dev_out, n * sizeof(hipFloatComplex), hipMemcpyDeviceToHost);    
    for (int i = 0; i < 20; ++i) {
        START_TIME;
        hipfftExecC2C(plan, dev_in, dev_out, direction);
        hipDeviceSynchronize();
        STOP_TIME(measures[i]);
    }
    hipfftDestroy(plan);
    hipFree(dev_in);
    hipFree(dev_out);
    return hipDeviceSynchronize();
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t FFT_ConstantGeometry(fftDirection direction, hipFloatComplex *in, hipFloatComplex *out, hipFloatComplex *W, double measures[], int n)
{
    unsigned int bufferswitch;
    hipFloatComplex *dev_in = 0;
    hipFloatComplex *dev_out = 0;
    hipFloatComplex *dev_W = 0;

    hipError_t cudaStatus = hipSetDevice(0);

    cudaStatus = hipMalloc((void**)&dev_in, n * sizeof(hipFloatComplex));
    cudaStatus = hipMalloc((void**)&dev_out, n * sizeof(hipFloatComplex));
    cudaStatus = hipMalloc((void**)&dev_W, (n / 2) * sizeof(hipFloatComplex));
    cudaStatus = hipMemcpy(dev_in, in, n * sizeof(hipFloatComplex), hipMemcpyHostToDevice);

    FFTConstGeom(direction, dev_in, dev_out, dev_W, &bufferswitch, n);
    hipDeviceSynchronize();
        
    cudaStatus = hipMemcpy(out, (bufferswitch == 1) ? dev_out : dev_in, n * sizeof(hipFloatComplex), hipMemcpyDeviceToHost);
    cudaStatus = hipMemcpy(in, (bufferswitch == 0) ? dev_out : dev_in, n * sizeof(hipFloatComplex), hipMemcpyDeviceToHost);
    cudaStatus = hipMemcpy(W, dev_W, (n / 2) * sizeof(hipFloatComplex), hipMemcpyDeviceToHost);

    for (int i = 0; i < 20; ++i) {
        START_TIME;
        FFTConstGeom(direction, dev_in, dev_out, dev_W, &bufferswitch, n);
        hipDeviceSynchronize();
        STOP_TIME(measures[i]);
    }

    hipFree(dev_in);
    hipFree(dev_out);
    hipFree(dev_W);

    return hipDeviceSynchronize();
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t FFT_ConstantGeometry2(fftDirection direction, hipFloatComplex *in, hipFloatComplex *out, double measures[], int n)
{
    unsigned int bufferswitch;
    hipFloatComplex *dev_in = 0;
    hipFloatComplex *dev_out = 0;
    hipError_t cudaStatus = hipSetDevice(0);
    cudaStatus = hipMalloc((void**)&dev_in, n * sizeof(hipFloatComplex));
    cudaStatus = hipMalloc((void**)&dev_out, n * sizeof(hipFloatComplex));
    cudaStatus = hipMemcpy(dev_in, in, n * sizeof(hipFloatComplex), hipMemcpyHostToDevice);
    FFTConstGeom2(direction, dev_in, dev_out, &bufferswitch, n);
    hipDeviceSynchronize();
    cudaStatus = hipMemcpy(out, (bufferswitch == 1) ? dev_out : dev_in, n * sizeof(hipFloatComplex), hipMemcpyDeviceToHost);
    cudaStatus = hipMemcpy(in, (bufferswitch == 0) ? dev_out : dev_in, n * sizeof(hipFloatComplex), hipMemcpyDeviceToHost);
    for (int i = 0; i < 20; ++i) {
        START_TIME;
        FFTConstGeom2(direction, dev_in, dev_out, &bufferswitch, n);
        hipDeviceSynchronize();
        STOP_TIME(measures[i]);
    }
    hipFree(dev_in);
    hipFree(dev_out);
    return hipDeviceSynchronize();
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t FFT_Tobb(fftDirection direction, hipFloatComplex *in, hipFloatComplex *out, double measures[], int n)
{
    unsigned int bufferswitch;
    hipFloatComplex *dev_in = 0;
    hipFloatComplex *dev_out = 0;
    hipFloatComplex *dev_W = 0;
    hipError_t cudaStatus = hipSetDevice(0);
    cudaStatus = hipMalloc((void**)&dev_in, n * sizeof(hipFloatComplex));
    cudaStatus = hipMalloc((void**)&dev_out, n * sizeof(hipFloatComplex));
    cudaStatus = hipMalloc((void**)&dev_W, (n / 2) * sizeof(hipFloatComplex));
    cudaStatus = hipMemcpy(dev_in, in, n * sizeof(hipFloatComplex), hipMemcpyHostToDevice);
    FFTTobb(direction, dev_in, dev_out, dev_W, &bufferswitch, n);
    hipDeviceSynchronize();
    cudaStatus = hipMemcpy(out, (bufferswitch == 1) ? dev_out : dev_in, n * sizeof(hipFloatComplex), hipMemcpyDeviceToHost);
    cudaStatus = hipMemcpy(in, (bufferswitch == 0) ? dev_out : dev_in, n * sizeof(hipFloatComplex), hipMemcpyDeviceToHost);
    for (int i = 0; i < 20; ++i) {
        START_TIME;
        FFTTobb(direction, dev_in, dev_out, dev_W, &bufferswitch, n);
        hipDeviceSynchronize();
        STOP_TIME(measures[i]);
    }
    hipFree(dev_in);
    hipFree(dev_out);
    hipFree(dev_W);
    return hipDeviceSynchronize();
}

/*
// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel << <1, size >> >(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);

    return cudaStatus;
}
*/
