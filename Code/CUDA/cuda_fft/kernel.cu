
#include <Windows.h>
#include "hip/hip_runtime.h"
#include ""
#include "math.h"

#include <stdio.h>

#include "definitions.cuh"
#include "fft_test.cuh"
#include "fft_const_geom.cuh"
#include "fft_helper.cuh"

#define NO_TESTS 32
#define MAX_LENGTH 2097152 / 2

/* Performance measure on Windows, result in micro seconds */

LARGE_INTEGER StartingTime, EndingTime, ElapsedMicroseconds, Frequency;
#define QPF QueryPerformanceFrequency
#define QPC QueryPerformanceCounter
#define START_TIME QPF(&Frequency); QPC(&StartingTime)
#define STOP_TIME(RES) QPC(&EndingTime); ElapsedMicroseconds.QuadPart = EndingTime.QuadPart - StartingTime.QuadPart; ElapsedMicroseconds.QuadPart *= 1000000; ElapsedMicroseconds.QuadPart /= Frequency.QuadPart;(RES) = (double)ElapsedMicroseconds.QuadPart

hipError_t fftCuda(float direction, cpx *in, cpx *out, int n);

int main()
{
    int n;
    double measures[2];
    cpx *in, *out, *ref;
    hipError_t cudaStatus;

    n = 8;
    in = get_seq(n, 1);
    ref = get_seq(n, in);
    out = get_seq(n);

    printf("Go go go!\n");
    getchar();

    printf("Forward\n");
    START_TIME;
    cudaStatus = fftCuda(-1.f, in, out, n);
    STOP_TIME(measures[0]);
    
    console_print(in, n);

    printf("Inverse\n");
    START_TIME;
    //cudaStatus = fftCuda(1.f, out, in, n);
    STOP_TIME(measures[1]);

    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        getchar();
        return 1;
    }

    printf("Happened: %f & %f\n", measures[0], measures[1]);
    checkError(in, ref, n, 1);

    free(in);
    free(out);
    free(ref);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        getchar();
        return 1;
    }

    getchar();
    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t fftCuda(float direction, cpx *in, cpx *out, int n)
{
    cpx *dev_in = 0;
    cpx *dev_out = 0;
    cpx *dev_W = 0;
    
    hipError_t cudaStatus;
        
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    printf("Got device!\n");

    cudaStatus = hipMalloc((void**)&dev_in, n * sizeof(cpx));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_out, n * sizeof(cpx));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_W, (n / 2) * sizeof(cpx));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_in, in, n * sizeof(cpx), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    fft_const_geom(direction, dev_in, dev_out, dev_W, n);
        
    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Last error: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(out, dev_out, n * sizeof(cpx), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_in);
    hipFree(dev_out);
    hipFree(dev_W);

    return cudaStatus;
}

/*
// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel << <1, size >> >(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);

    return cudaStatus;
}
*/
