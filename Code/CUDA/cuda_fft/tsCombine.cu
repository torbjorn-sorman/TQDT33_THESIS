#include "hip/hip_runtime.h"
#include "tsCombine.cuh"

__global__ void _kernelAll(cpx *in, cpx *out, const float angle, const unsigned int lmask, const unsigned int pmask, const int steps, const int dist);
__global__ void _kernelBlock(cpx *in, cpx *out, const float angle, const int depth, const int n2);
__global__ void _kernelB(cpx *in, cpx *out, const float angle, const cpx scale, const int depth, const unsigned int lead, const int n2);

__host__ int tsCombine_Validate(const int n)
{
    cpx *in, *ref, *out, *dev_in, *dev_out;
    fftMalloc(n, &dev_in, &dev_out, NULL, &in, &ref, &out);
    int mblock = (n > MAX_BLOCK_SIZE * 2) && 1;
    
    if (mblock) {
        printf("\nIN:\n");
        console_print(in, n);
        printf("\n");
    }
    hipMemcpy(dev_in, in, n * sizeof(cpx), hipMemcpyHostToDevice);
    tsCombine(FFT_FORWARD, &dev_in, &dev_out, n);

    if (mblock) {
        hipMemcpy(out, dev_out, n * sizeof(cpx), hipMemcpyDeviceToHost);
        printf("\nOUT:\n");
        console_print(out, n);
        printf("\n");
    }

    tsCombine(FFT_INVERSE, &dev_out, &dev_in, n);
    hipMemcpy(in, dev_in, n * sizeof(cpx), hipMemcpyDeviceToHost);
    /*
    if (mblock) {
        printf("\n");
        console_print(in, 10);
        printf("\n");
    }
    */
    return fftResultAndFree(n, &dev_in, &dev_out, NULL, &in, &ref, &out) != 1;
}

__host__ double tsCombine_Performance(const int n)
{
    double measures[NUM_PERFORMANCE];
    cpx *in, *ref, *out, *dev_in, *dev_out;
    fftMalloc(n, &dev_in, &dev_out, NULL, &in, &ref, &out);

    hipMemcpy(dev_in, in, n * sizeof(cpx), hipMemcpyHostToDevice);
    for (int i = 0; i < NUM_PERFORMANCE; ++i) {
        startTimer();
        tsCombine(FFT_FORWARD, &dev_in, &dev_out, n);
        measures[i] = stopTimer();
    }

    fftResultAndFree(n, &dev_in, &dev_out, NULL, &in, &ref, &out);
    return avg(measures, NUM_PERFORMANCE);
}

__host__ void tsCombine(fftDirection dir, cpx **dev_in, cpx **dev_out, const int n)
{
    int threadsPerBlock, numBlocks;
    const float w_angle = dir * (M_2_PI / n);
    const int depth = log2_32(n);
    const int n2 = (n / 2);
    const int breakSize = log2_32(MAX_BLOCK_SIZE);
    int steps = 0;
    int bit = depth - 1;
    int dist = n2;

    // Set number of blocks and threads
    setBlocksAndThreads(&numBlocks, &threadsPerBlock, n2);
    if (numBlocks > 1) {
        //printf("lvl 1\n");
        const float scale = dir == FFT_FORWARD ? 1.f : 1.f / n;
        // Sync at device level until 
        _kernelAll KERNEL_ARGS2(numBlocks, threadsPerBlock)(*dev_in, *dev_out, w_angle, 0xFFFFFFFF << bit, (dist - 1) << steps, steps, dist);
        hipDeviceSynchronize();
        while (bit-- > breakSize) {
            dist = dist >> 1;
            ++steps;
            _kernelAll KERNEL_ARGS2(numBlocks, threadsPerBlock)(*dev_out, *dev_out, w_angle, 0xFFFFFFFF << bit, (dist - 1) << steps, steps, dist);
            hipDeviceSynchronize();
        }
        const int nBlock = n / numBlocks;
        printf("\nblocks: %d\tang: %f\tbit: %d\tn2: %d\n", numBlocks, dir * (M_2_PI / nBlock), log2_32(nBlock), nBlock / 2);
        
        cpx *out = (cpx *)malloc(sizeof(cpx) * n);
        hipMemcpy(out, *dev_out, n * sizeof(cpx), hipMemcpyDeviceToHost);
        console_print(out, n);


        _kernelBlock KERNEL_ARGS3(numBlocks, threadsPerBlock, sizeof(cpx) * nBlock)(*dev_out, *dev_in, dir * (M_2_PI / nBlock), log2_32(nBlock), nBlock / 2);
        hipDeviceSynchronize();

        setBlocksAndThreads(&numBlocks, &threadsPerBlock, n);
        bit_reverse KERNEL_ARGS2(numBlocks, threadsPerBlock)(*dev_in, *dev_out, scale, 32 - depth);      
    }
    else {
        const cpx scale = make_hipFloatComplex((dir == FFT_FORWARD ? 1.f : 1.f / n), 0.f);
        _kernelB KERNEL_ARGS3(1, threadsPerBlock, sizeof(cpx) * n)(*dev_in, *dev_out, w_angle, scale, depth, 32 - depth, n2);
    }
    hipDeviceSynchronize();
}

// Take no usage of shared mem yet...
__global__ void _kernelAll(cpx *in, cpx *out, const float angle, const unsigned int lmask, const unsigned int pmask, const int steps, const int dist)
{
    int tid = (blockIdx.x * blockDim.x + threadIdx.x);
    int l = tid + (tid & lmask);
    int u = l + dist;
    cpx in_lower = in[l];
    cpx in_upper = in[u];
    cpx w;
    SIN_COS_F(angle * ((tid << steps) & pmask), &w.y, &w.x);
    cpx_add_sub_mul(&(out[l]), &(out[u]), in_lower, in_upper, w);
}

// Full usage of shared mem!
__global__ void _kernelBlock(cpx *in, cpx *out, const float angle, const int depth, const int n2)
{
    extern __shared__ cpx shared[];
    cpx w, in_lower, in_upper;
    const int tid = (blockIdx.x * blockDim.x + threadIdx.x);
    const int in_low = threadIdx.x;
    const int in_high = n2 / 2 + in_low;
    const int i = (in_low << 1);
    const int ii = i + 1;
        
    /* Move Global to Shared */
    shared[in_low] = in[in_low + blockIdx.x * blockDim.x];
    shared[in_high] = in[in_high + blockIdx.x * blockDim.x];

    if (tid == n2 || tid == 0) {
        printf("%d\t(%d, %d) -> (%d, %d)\t%d\t%d\n", tid, in_low, in_high, i, ii, blockIdx.x, blockIdx.x * blockDim.x);
    }

    /* Run FFT algorithm */
    for (int steps = 0; steps < depth; ++steps) {
        SYNC_THREADS;
        in_lower = shared[in_low];
        in_upper = shared[in_high];
        SIN_COS_F(angle * ((in_low & (0xffffffff << steps))), &w.y, &w.x);
        SYNC_THREADS;
        cpx_add_sub_mul(&(shared[i]), &(shared[ii]), in_lower, in_upper, w);
    }

    /* Move Shared to Global */
    SYNC_THREADS;    
    out[in_low + blockIdx.x * blockDim.x] = shared[in_low];
    out[in_high + blockIdx.x * blockDim.x] = shared[in_high];
}

__global__ void _kernelB(cpx *in, cpx *out, const float angle, const cpx scale, const int depth, const unsigned int lead, const int n2)
{
    extern __shared__ cpx shared[];
    cpx w, in_lower, in_upper;
    const int tid = (blockIdx.x * blockDim.x + threadIdx.x);
    const int in_high = n2 + tid;
    const int i = (tid << 1);
    const int ii = i + 1;

    /* Move (bit-reversed?) Global to Shared */
    globalToShared(tid, in_high, 0, lead, shared, in);

    /* Run FFT algorithm */
    for (int steps = 0; steps < depth; ++steps) {
        SYNC_THREADS;
        in_lower = shared[tid];
        in_upper = shared[in_high];
        SIN_COS_F(angle * ((tid & (0xffffffff << steps))), &w.y, &w.x);
        SYNC_THREADS;
        cpx_add_sub_mul(&(shared[i]), &(shared[ii]), in_lower, in_upper, w);
    }

    /* Move Shared to Global (index bit-reversed) */
    SYNC_THREADS;
    sharedToGlobal(tid, in_high, 0, scale, lead, shared, out);
}