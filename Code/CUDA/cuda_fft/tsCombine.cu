#include "hip/hip_runtime.h"
#include "tsCombine.cuh"

typedef int syncVal;

__global__ void _kernelAll(cpx *in, cpx *out, float angle, unsigned int lmask, int steps, int dist);
__global__ void _kernelAll2DRow(cpx *in, cpx *out, float angle, unsigned int lmask, int steps, int dist);
__global__ void _kernelAll2DCol(cpx *in, cpx *out, float angle, unsigned int lmask, int steps, int dist);
__global__ void _kernelGPUS(cpx *in, cpx *out, float angle, float bAngle, int depth, int lead, int breakSize, cpx scale, int nBlocks, int n2);
__global__ void _kernelGPUS2DRow(cpx *in, cpx *out, float angle, float bAngle, int depth, cpx scale, int n);
__global__ void _kernelGPUS2DCol(cpx *in, cpx *out, float angle, float bAngle, int depth, cpx scale, int n);

__host__ int tsCombine_Validate(int n)
{
    cpx *in, *ref, *out, *dev_in, *dev_out;
    fftMalloc(n, &dev_in, &dev_out, NULL, &in, &ref, &out);
    hipMemcpy(dev_in, in, n * sizeof(cpx), hipMemcpyHostToDevice);
    tsCombine(FFT_FORWARD, &dev_in, &dev_out, n);
    tsCombine(FFT_INVERSE, &dev_out, &dev_in, n);
    hipMemcpy(in, dev_in, n * sizeof(cpx), hipMemcpyDeviceToHost);

    return fftResultAndFree(n, &dev_in, &dev_out, NULL, &in, &ref, &out) != 1;
}

__host__ void testCombine2DRun(fftDir dir, cpx *in, cpx **dev_in, cpx **dev_out, char *image_name, char *type, size_t size, int write, int norm, int n)
{
    tsCombine2D(dir, dev_in, dev_out, n);
    if (write) {
        hipMemcpy(in, *dev_out, size, hipMemcpyDeviceToHost);
        if (norm) {
            normalized_image(in, n);
            cpx *tmp = fftShift(in, n);
            write_image(image_name, type, tmp, n);
            free(tmp);
        }
        else {
            write_image(image_name, type, in, n);
        }
    }
}

__host__ int tsCombine2D_Validate(int n)
{
    char *image_name = "shore";
    cpx *in, *ref, *dev_in, *dev_out;
    size_t size;
    fft2DSetup(&in, &ref, &dev_in, &dev_out, &size, image_name, 0, n);

    //hipMemcpy(dev_in, in, size, hipMemcpyHostToDevice);
    //tsCombine2D(FFT_FORWARD, &dev_in, &dev_out, n);
    //tsCombine2D(FFT_INVERSE, &dev_out, &dev_in, n);

    hipMemcpy(dev_in, in, size, hipMemcpyHostToDevice);
    testCombine2DRun(FFT_FORWARD, in, &dev_in, &dev_out, image_name, "frequency-domain", size, 1, 1, n);
    testCombine2DRun(FFT_INVERSE, in, &dev_out, &dev_in, image_name, "spatial-domain", size, 1, 0, n);

    int res = fft2DCompare(in, ref, dev_in, size, n * n);
    fft2DShakedown(&in, &ref, &dev_in, &dev_out);
    return res;
}

__host__ double tsCombine_Performance(int n)
{
    double measures[NUM_PERFORMANCE];
    cpx *in, *ref, *out, *dev_in, *dev_out;
    fftMalloc(n, &dev_in, &dev_out, NULL, &in, &ref, &out);

    hipMemcpy(dev_in, in, n * sizeof(cpx), hipMemcpyHostToDevice);
    for (int i = 0; i < NUM_PERFORMANCE; ++i) {
        startTimer();
        tsCombine(FFT_FORWARD, &dev_in, &dev_out, n);
        measures[i] = stopTimer();
    }

    fftResultAndFree(n, &dev_in, &dev_out, NULL, &in, &ref, &out);
    return avg(measures, NUM_PERFORMANCE);
}

__host__ double tsCombine2D_Performance(int n)
{
    double measures[NUM_PERFORMANCE];
    char *image_name = "splash";
    cpx *in, *ref, *dev_in, *dev_out;
    size_t size;
    fft2DSetup(&in, &ref, &dev_in, &dev_out, &size, image_name, 0, n);

    for (int i = 0; i < NUM_PERFORMANCE; ++i) {
        startTimer();
        tsCombine2D(FFT_FORWARD, &dev_in, &dev_out, n);
        measures[i] = stopTimer();
    }

    fft2DShakedown(&in, &ref, &dev_in, &dev_out);
    return avg(measures, NUM_PERFORMANCE);
}

// My device specifics!
// Seven physical "cores" that can run blocks in "parallel" (and most important sync threads in a block). 1024 is the thread limit per physical core.
// Essentially (depending on scheduling and other factors) # blocks fewer than HW_LIMIT can be synched, any # above is not trivially solved. cuFFT solves this.
#define HW_LIMIT (1024 / MAX_BLOCK_SIZE) * 7

__host__ void tsCombine(fftDir dir, cpx **dev_in, cpx **dev_out, int n)
{
    int threads, blocks;
    int depth = log2_32(n);
    const int lead = 32 - depth;
    const int n2 = (n / 2);
    const int breakSize = log2_32(MAX_BLOCK_SIZE);
    cpx scaleCpx = make_hipFloatComplex((dir == FFT_FORWARD ? 1.f : 1.f / n), 0.f);
    set_block_and_threads(&blocks, &threads, n2);
    int numBlocks = blocks;
    const int nBlock = n / blocks;
    const float w_angle = dir * (M_2_PI / n);
    const float w_bangle = dir * (M_2_PI / nBlock);
    int bSize = n2;

    if (blocks >= HW_LIMIT) {
        // Calculate sequence until parts fit into a block, syncronize on CPU until then.
        --depth;
        int steps = 0;
        int dist = n2;
        _kernelAll KERNEL_ARGS2(blocks, threads)(*dev_in, *dev_out, w_angle, 0xFFFFFFFF << depth, steps, dist);
        hipDeviceSynchronize();
        while (--depth > breakSize) {
            dist >>= 1;
            ++steps;
            _kernelAll KERNEL_ARGS2(blocks, threads)(*dev_out, *dev_out, w_angle, 0xFFFFFFFF << depth, steps, dist);
            hipDeviceSynchronize();
        }
        swap(dev_in, dev_out);
        ++depth;
        bSize = nBlock / 2;
        numBlocks = 1;
    }

    // Calculate complete sequence in one launch and syncronize on GPU
    _kernelGPUS KERNEL_ARGS3(blocks, threads, sizeof(cpx) * nBlock) (*dev_in, *dev_out, w_angle, w_bangle, depth, lead, breakSize, scaleCpx, numBlocks, bSize);
    hipDeviceSynchronize();
}

#define ROW_COL_KERNEL(rw, kr, kc) ((rw) ? (kr) : (kc))

__host__ void tsCombine2D_help(fftDir dir, cpx **dev_in, cpx **dev_out, int rowWise, int n)
{
    dim3 blocks;
    int threads;
    int depth = log2_32(n);
    const int breakSize = log2_32(MAX_BLOCK_SIZE);
    cpx scaleCpx = make_hipFloatComplex((dir == FFT_FORWARD ? 1.f : 1.f / n), 0.f);
    set_block_and_threads2D(&blocks, &threads, n);

    const int nBlock = n / blocks.y;
    const float w_angle = dir * (M_2_PI / n);
    const float w_bangle = dir * (M_2_PI / nBlock);
    int bSize = n;
    if (blocks.y > 1) {

        // Calculate sequence until parts fit into a block, syncronize on CPU until then.
        --depth;
        int steps = 0;
        int dist = (n / 2);
        ROW_COL_KERNEL(rowWise, _kernelAll2DRow, _kernelAll2DCol) KERNEL_ARGS2(blocks, threads)(*dev_in, *dev_out, w_angle, 0xFFFFFFFF << depth, steps, dist);
        hipDeviceSynchronize();
        while (--depth > breakSize) {
            dist >>= 1;
            ++steps;
            ROW_COL_KERNEL(rowWise, _kernelAll2DRow, _kernelAll2DCol) KERNEL_ARGS2(blocks, threads)(*dev_out, *dev_out, w_angle, 0xFFFFFFFF << depth, steps, dist);
            hipDeviceSynchronize();
        }
        swap(dev_in, dev_out);
        ++depth;
        bSize = nBlock;
    }

    // Calculate complete sequence in one launch and syncronize on GPU
    ROW_COL_KERNEL(rowWise, _kernelGPUS2DRow, _kernelGPUS2DCol) KERNEL_ARGS3(blocks, threads, sizeof(cpx) * nBlock) (*dev_in, *dev_out, w_angle, w_bangle, depth, scaleCpx, bSize);
    hipDeviceSynchronize();
}

__host__ void tsCombine2D(fftDir dir, cpx **dev_in, cpx **dev_out, int n)
{
    dim3 blocks, threads;
    set_block_and_threads_transpose(&blocks, &threads, n);

    if (n > 256) {
        tsCombine2D_help(dir, dev_in, dev_out, 1, n);
        _kernelTranspose KERNEL_ARGS2(blocks, threads) (*dev_out, *dev_in, n);
        hipDeviceSynchronize();
        tsCombine2D_help(dir, dev_in, dev_out, 1, n);
        _kernelTranspose KERNEL_ARGS2(blocks, threads) (*dev_out, *dev_in, n);
        hipDeviceSynchronize();
    }
    else {
        tsCombine2D_help(dir, dev_in, dev_out, 1, n);
        tsCombine2D_help(dir, dev_out, dev_in, 0, n);
    }
    swap(dev_in, dev_out);
}

__device__ static __inline__ void inner_k(cpx *in, cpx *out, float angle, int steps, unsigned int lmask, int dist)
{
    cpx w;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int in_low = tid + (tid & lmask);
    int in_high = in_low + dist;
    cpx in_lower = in[in_low];
    cpx in_upper = in[in_high];
    SIN_COS_F(angle * ((tid << steps) & ((dist - 1) << steps)), &w.y, &w.x);
    cpx_add_sub_mul(&(out[in_low]), &(out[in_high]), in_lower, in_upper, w);
}

__device__ static __inline__ int algorithm_c(cpx *in, cpx *out, int bit_start, int breakSize, float angle, int nBlocks, int n2)
{
    int tid = (blockIdx.x * blockDim.x + threadIdx.x);
    int dist = n2;
    int steps = 0;
    init_sync(tid, nBlocks);
    inner_k(in, out, angle, steps, 0xFFFFFFFF << bit_start, dist);
    __gpu_sync(nBlocks + steps);
    for (int bit = bit_start - 1; bit > breakSize; --bit) {
        dist = dist >> 1;
        ++steps;
        inner_k(out, out, angle, steps, 0xFFFFFFFF << bit, dist);
        __gpu_sync(nBlocks + steps);
    }
    return breakSize + 1;
}

__device__ static __inline__ void algorithm_p(cpx *shared, int in_high, float angle, int bit)
{
    cpx w, in_lower, in_upper;
    int i = (threadIdx.x << 1);
    int ii = i + 1;
    for (int steps = 0; steps < bit; ++steps) {
        in_lower = shared[threadIdx.x];
        in_upper = shared[in_high];
        SYNC_THREADS;
        SIN_COS_F(angle * ((threadIdx.x & (0xFFFFFFFF << steps))), &w.y, &w.x);
        cpx_add_sub_mul(&(shared[i]), &(shared[ii]), in_lower, in_upper, w);
        SYNC_THREADS;
    }
}

// Take no usage of shared mem yet...
__global__ void _kernelAll(cpx *in, cpx *out, float angle, unsigned int lmask, int steps, int dist)
{
    inner_k(in, out, angle, steps, lmask, dist);
}

__device__ static __inline__ void inner_k2D(cpx *in, cpx *out, int x, int y, float angle, int steps, int dist)
{
    cpx w;
    int tid = (blockIdx.y * blockDim.x + threadIdx.x);
    int in_low = x + y * gridDim.x;
    int in_high = in_low + dist;
    cpx in_lower = in[in_low];
    cpx in_upper = in[in_high];
    SIN_COS_F(angle * ((tid << steps) & ((dist - 1) << steps)), &w.y, &w.x);
    cpx_add_sub_mul(&(out[in_low]), &(out[in_high]), in_lower, in_upper, w);
}

__global__ void _kernelAll2DRow(cpx *in, cpx *out, float angle, unsigned int lmask, int steps, int dist)
{
    int col_id = blockIdx.y * blockDim.x + threadIdx.x;
    inner_k2D(in, out, (col_id + (col_id & lmask)), blockIdx.x, angle, steps, dist);
}

__global__ void _kernelAll2DCol(cpx *in, cpx *out, float angle, unsigned int lmask, int steps, int dist)
{
    int row_id = blockIdx.y * blockDim.x + threadIdx.x;
    inner_k2D(in, out, blockIdx.x, (row_id + (row_id & lmask)), angle, steps, dist);
}

// Full blown block syncronized algorithm! In theory this should scale up but is limited by hardware (#cores)
__global__ void _kernelGPUS(cpx *in, cpx *out, float angle, float bAngle, int depth, int lead, int breakSize, cpx scale, int nBlocks, int n2)
{
    extern __shared__ cpx shared[];
    int bit = depth;
    int in_high = n2;
    if (nBlocks > 1) {
        bit = algorithm_c(in, out, depth - 1, breakSize, angle, nBlocks, in_high);
        in_high >>= log2(nBlocks);
        in = out;
    }
    int offset = blockIdx.x * blockDim.x * 2;
    in_high += threadIdx.x;
    mem_gtos(threadIdx.x, in_high, offset, shared, in);
    SYNC_THREADS;
    algorithm_p(shared, in_high, bAngle, bit);
    mem_stog_db(threadIdx.x, in_high, offset, lead, scale, shared, out);
}

__global__ void _kernelGPUS2DRow(cpx *in, cpx *out, float angle, float bAngle, int depth, cpx scale, int nBlock)
{
    extern __shared__ cpx shared[];
    int rowStart = gridDim.x * blockIdx.x;
    int in_high = (nBlock >> 1) + threadIdx.x;
    int rowOffset = blockIdx.y * blockDim.x * 2;
    mem_gtos(threadIdx.x, in_high, rowOffset, shared, &(in[rowStart]));
    SYNC_THREADS;
    algorithm_p(shared, in_high, bAngle, depth);
    mem_stog_db(threadIdx.x, in_high, rowOffset, (32 - log2((int)gridDim.x)), scale, shared, &(out[rowStart]));
}

__global__ void _kernelGPUS2DCol(cpx *in, cpx *out, float angle, float bAngle, int depth, cpx scale, int n)
{
    extern __shared__ cpx shared[];
    int in_high = n >> 1;
    int colOffset = blockIdx.y * blockDim.x * 2;
    in_high += threadIdx.x;
    mem_gtos_col(threadIdx.x, in_high, (threadIdx.x + colOffset) * n + blockIdx.x, (n >> 1) * n, shared, in);
    SYNC_THREADS;
    algorithm_p(shared, in_high, bAngle, depth);
    mem_stog_db_col(threadIdx.x, in_high, colOffset, 32 - log2((int)gridDim.x), scale, shared, out, n);
}

// ---------------------------------------------
//
// Surface / Texture object
//
// Experimental so see if it scales up better then using global memory
//
// ---------------------------------------------

__device__ static __inline__ void inner_k2DRowSurf(cuSurf in, cuSurf out, int x, int y, float angle, int steps, int dist)
{
    cpx w, in_lower, in_upper;
    SURF2D_READ(&in_lower, in, x, y);
    SURF2D_READ(&in_upper, in, x + dist, y);
    SIN_COS_F(angle * (((blockIdx.y * blockDim.x + threadIdx.x) << steps) & ((dist - 1) << steps)), &w.y, &w.x);
    SURF2D_WRITE(hipCaddf(in_lower, in_upper), out, x, y);
    SURF2D_WRITE(hipCmulf(hipCsubf(in_lower, in_upper), w), out, x + dist, y);
}
__device__ static __inline__ void inner_k2DColSurf(cuSurf in, cuSurf out, int x, int y, float angle, int steps, int dist)
{
    cpx w, in_lower, in_upper;
    SURF2D_READ(&in_lower, in, x, y);
    SURF2D_READ(&in_upper, in, x, y + dist);
    SIN_COS_F(angle * (((blockIdx.y * blockDim.x + threadIdx.x) << steps) & ((dist - 1) << steps)), &w.y, &w.x);
    SURF2D_WRITE(hipCaddf(in_lower, in_upper), out, x, y);
    SURF2D_WRITE(hipCmulf(hipCsubf(in_lower, in_upper), w), out, x, y + dist);
}

__global__ void _kernelAll2DRowSurf(cuSurf in, cuSurf out, float angle, unsigned int lmask, int steps, int dist)
{
    int col_id = blockIdx.y * blockDim.x + threadIdx.x;
    inner_k2DRowSurf(in, out, (col_id + (col_id & lmask)), blockIdx.x, angle, steps, dist);
}

__global__ void _kernelAll2DColSurf(cuSurf in, cuSurf out, float angle, unsigned int lmask, int steps, int dist)
{
    int row_id = blockIdx.y * blockDim.x + threadIdx.x;
    inner_k2DColSurf(in, out, blockIdx.x, (row_id + (row_id & lmask)), angle, steps, dist);
}

__global__ void _kernelGPUS2DRowSurf(cuSurf in, cuSurf out, float angle, float bAngle, int depth, cpx scale, int nBlock)
{
    extern __shared__ cpx shared[];
    int in_high = (nBlock >> 1) + threadIdx.x;
    int rowOffset = blockIdx.y * blockDim.x * 2;
    mem_gtos_row(threadIdx.x, in_high, rowOffset, shared, in);
    SYNC_THREADS;
    algorithm_p(shared, in_high, bAngle, depth);
    mem_stog_db_row(threadIdx.x, in_high, rowOffset, (32 - log2((int)gridDim.x)), scale, shared, out);
}

__global__ void _kernelGPUS2DColSurf(cuSurf in, cuSurf out, float angle, float bAngle, int depth, cpx scale, int n)
{
    extern __shared__ cpx shared[];
    int in_high = n >> 1;
    int colOffset = blockIdx.y * blockDim.x * 2;
    in_high += threadIdx.x;
    mem_gtos_col((int)threadIdx.x, in_high, colOffset, shared, in);
    SYNC_THREADS;
    algorithm_p(shared, in_high, bAngle, depth);
    mem_stog_db_col(threadIdx.x, in_high, colOffset, 32 - log2((int)gridDim.x), scale, shared, out);
}

__host__ void tsCombine2DSurf_help(fftDir dir, cuSurf *surfaceIn, cuSurf *surfaceOut, int rowWise, int n)
{
    dim3 blocks;
    int threads;
    int depth = log2_32(n);
    const int breakSize = log2_32(MAX_BLOCK_SIZE);
    cpx scaleCpx = make_hipFloatComplex((dir == FFT_FORWARD ? 1.f : 1.f / n), 0.f);
    set_block_and_threads2D(&blocks, &threads, n);

    const int nBlock = n / blocks.y;
    const float w_angle = dir * (M_2_PI / n);
    const float w_bangle = dir * (M_2_PI / nBlock);
    int bSize = n;
    if (blocks.y > 1) {

        // Calculate sequence until parts fit into a block, syncronize on CPU until then.
        --depth;
        int steps = 0;
        int dist = (n / 2);
        ROW_COL_KERNEL(rowWise, _kernelAll2DRowSurf, _kernelAll2DColSurf) KERNEL_ARGS2(blocks, threads)(*surfaceIn, *surfaceOut, w_angle, 0xFFFFFFFF << depth, steps, dist);
        hipDeviceSynchronize();
        while (--depth > breakSize) {
            dist >>= 1;
            ++steps;
            ROW_COL_KERNEL(rowWise, _kernelAll2DRowSurf, _kernelAll2DColSurf) KERNEL_ARGS2(blocks, threads)(*surfaceOut, *surfaceOut, w_angle, 0xFFFFFFFF << depth, steps, dist);
            hipDeviceSynchronize();
        }
        swap(surfaceIn, surfaceOut);
        ++depth;
        bSize = nBlock;
    }
    // Calculate complete sequence in one launch and syncronize on GPU
    ROW_COL_KERNEL(rowWise, _kernelGPUS2DRowSurf, _kernelGPUS2DColSurf) KERNEL_ARGS3(blocks, threads, sizeof(cpx) * nBlock) (*surfaceIn, *surfaceOut, w_angle, w_bangle, depth, scaleCpx, bSize);
    hipDeviceSynchronize();
}

__host__ void tsCombine2DSurf(fftDir dir, cuSurf *surfaceIn, cuSurf *surfaceOut, int n)
{
    dim3 blocks, threads;
    set_block_and_threads_transpose(&blocks, &threads, n);

    tsCombine2DSurf_help(dir, surfaceIn, surfaceOut, 1, n);
    tsCombine2DSurf_help(dir, surfaceOut, surfaceIn, 0, n);

    swap(surfaceIn, surfaceOut);
}

__host__ void _testTex2DShakedown(cpx **in, cpx **ref, cuSurf *sObjIn, cuSurf *sObjOut, hipArray **cuAIn, hipArray **cuAOut)
{
    free(*in);
    free(*ref);
    hipDestroySurfaceObject(*sObjIn);
    hipFreeArray(*cuAIn);
    if (sObjOut != NULL) {
        hipDestroySurfaceObject(*sObjOut);
        hipFreeArray(*cuAOut);
    }
}

__host__ void _testTex2DRun(fftDir dir, cpx *in, hipArray *dev, cuSurf *surfIn, cuSurf *surfOut, char *image_name, char *type, size_t size, int write, int norm, int n)
{
    tsCombine2DSurf(dir, surfIn, surfOut, n);
    if (write) {
        hipMemcpyFromArray(in, dev, 0, 0, size, hipMemcpyDeviceToHost);
        if (norm) {
            normalized_image(in, n);
            cpx *tmp = fftShift(in, n);
            write_image(image_name, type, tmp, n);
            free(tmp);
        }
        else {
            write_image(image_name, type, in, n);
        }
    }
}

__host__ int _testTex2DCompare(cpx *in, cpx *ref, hipArray *dev, size_t size, int len)
{
    hipMemcpyFromArray(in, dev, 0, 0, size, hipMemcpyDeviceToHost);
    for (int i = 0; i < len; ++i) {
        if (hipCabsf(hipCsubf(in[i], ref[i])) > 0.0001) {
            return 0;
        }
    }
    return 1;
}

#define YES 1
#define NO 0

__host__ int tsCombine2DSurf_Validate(int n)
{
    int res;
    char *image_name = "shore";
    cpx *in, *ref;
    size_t size;    
    hipArray *inArr, *outArr;
    cuSurf inSurf, outSurf;
    fft2DSurfSetup(&in, &ref, &size, image_name, NO, n, &inArr, &outArr, &inSurf, &outSurf);
    hipMemcpyToArray(inArr, 0, 0, in, size, hipMemcpyHostToDevice);
    _testTex2DRun(FFT_FORWARD, in, inArr, &inSurf, &outSurf, image_name, "surf-frequency-domain", size, YES, YES, n);
    _testTex2DRun(FFT_INVERSE, in, inArr, &outSurf, &inSurf, image_name, "surf-spatial-domain", size, YES, NO, n);
    res = _testTex2DCompare(in, ref, inArr, size, n * n);
    _testTex2DShakedown(&in, &ref, &inSurf, &outSurf, &inArr, &outArr);    
    return res;
}

__host__ double tsCombine2DSurf_Performance(int n)
{
    double measures[NUM_PERFORMANCE];
    char *image_name = "shore";
    cpx *in, *ref;
    size_t size;
    hipArray *inArr, *outArr;
    cuSurf inSurf, outSurf;
    fft2DSurfSetup(&in, &ref, &size, image_name, NO, n, &inArr, &outArr, &inSurf, &outSurf);

    for (int i = 0; i < NUM_PERFORMANCE; ++i) {
        startTimer();
        tsCombine2DSurf(FFT_FORWARD, &inSurf, &outSurf, n);
        cudaCheckError();
        measures[i] = stopTimer();
    }
    _testTex2DShakedown(&in, &ref, &inSurf, &outSurf, &inArr, &outArr);
    return avg(measures, NUM_PERFORMANCE);
}