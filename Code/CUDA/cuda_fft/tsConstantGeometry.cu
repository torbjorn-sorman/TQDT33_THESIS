#include "hip/hip_runtime.h"
#include <stdio.h>
#include <>

#include "math.h"
#include "tsConstantGeometry.cuh"
#include "tsHelper.cuh"
#include "tsTest.cuh"

__global__ void _tsConstantGeometry_Body(cpx *in, cpx *out, cpx *W, unsigned int mask, const int n2);

__host__ int tsConstantGeometry_Validate(const size_t n)
{
    int result;
    cpx *in, *ref, *out, *dev_in, *dev_out, *dev_W;
    fftMalloc(n, &dev_in, &dev_out, &dev_W, &in, &ref, &out);

    hipMemcpy(dev_in, in, n * sizeof(cpx), hipMemcpyHostToDevice);
    tsConstantGeometry(FFT_FORWARD, &dev_in, &dev_out, dev_W, n);
    tsConstantGeometry(FFT_INVERSE, &dev_out, &dev_in, dev_W, n);    
    hipMemcpy(in, dev_in, n * sizeof(cpx), hipMemcpyDeviceToHost);

    return fftResultAndFree(n, &dev_in, &dev_out, &dev_W, &in, &ref, &out) != 1;
}

__host__ double tsConstantGeometry_Performance(const size_t n)
{
    double measures[NUM_PERFORMANCE];
    cpx *in, *ref, *out, *dev_in, *dev_out, *dev_W;
    fftMalloc(n, &dev_in, &dev_out, &dev_W, &in, &ref, &out);

    hipMemcpy(dev_in, in, n * sizeof(cpx), hipMemcpyHostToDevice);
    for (int i = 0; i < NUM_PERFORMANCE; ++i) {
        startTimer();
        tsConstantGeometry(FFT_FORWARD, &dev_in, &dev_out, dev_W, n);
        measures[i] = stopTimer();
    }

    fftResultAndFree(n, &dev_in, &dev_out, &dev_W, &in, &ref, &out);
    return avg(measures, NUM_PERFORMANCE);
}

__host__ void tsConstantGeometry(fftDirection dir, cpx **dev_in, cpx **dev_out, cpx *dev_W, const int n)
{
    int steps, depth, threadsPerBlock, numBlocks;
    const float w_angle = dir * (M_2_PI / n);
    const float scale = dir == FFT_FORWARD ? 1.f : 1.f / n;
    const int n2 = (n / 2);

    depth = log2_32(n);

    _setBlocksAndThreads(&numBlocks, &threadsPerBlock, n2);
    twiddle_factors KERNEL_ARGS2(numBlocks, threadsPerBlock)(dev_W, w_angle, n);
    hipDeviceSynchronize();
    
    steps = 0;
    _tsConstantGeometry_Body KERNEL_ARGS2(numBlocks, threadsPerBlock)(*dev_in, *dev_out, dev_W, 0xffffffff << steps, n2);
    hipDeviceSynchronize();
    while (++steps < depth) {
        swap(dev_in, dev_out);        
        _tsConstantGeometry_Body KERNEL_ARGS2(numBlocks, threadsPerBlock)(*dev_in, *dev_out, dev_W, 0xffffffff << steps, n2);
        hipDeviceSynchronize();
    }

    _setBlocksAndThreads(&numBlocks, &threadsPerBlock, n);    
    bit_reverse KERNEL_ARGS2(numBlocks, threadsPerBlock)(*dev_out, *dev_in, scale, 32 - depth);
    swap(dev_in, dev_out);
    hipDeviceSynchronize();
}

__global__ void _tsConstantGeometry_Body(cpx *in, cpx *out, cpx *W, unsigned int mask, const int n2)
{
    //__shared__ cpx input[256];
    int l = (blockIdx.x * blockDim.x + threadIdx.x);
    int i = l * 2;
    cpx in_lower = in[l];
    cpx in_upper = in[n2 + l];
    cpx twiddle = W[l & mask];
    out[i] = hipCaddf(in_lower, in_upper);
    out[i + 1] = hipCmulf(twiddle, hipCsubf(in_lower, in_upper));
}