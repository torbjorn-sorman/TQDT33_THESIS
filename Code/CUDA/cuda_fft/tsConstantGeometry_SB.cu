#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <>
#include <hip/device_functions.h>

#include "math.h"
#include "tsConstantGeometry_SB.cuh"
#include "tsHelper.cuh"
#include "tsTest.cuh"

__global__ void _kernelCGSB(cpx *in, cpx *out, const float angle, const cpx scale, int depth, unsigned int lead, const int n2);
__global__ void _kernelCGSB48K(cpx *in, cpx *out, const float angle, const cpx scale, int depth, unsigned int lead, const int n2);

__host__ int tsConstantGeometry_SB_Validate(const int n)
{
    cpx *in, *ref, *out, *dev_in, *dev_out;
    fftMalloc(n, &dev_in, &dev_out, NULL, &in, &ref, &out);

    hipMemcpy(dev_in, in, n * sizeof(cpx), hipMemcpyHostToDevice);
    tsConstantGeometry_SB(FFT_FORWARD, &dev_in, &dev_out, n);
    tsConstantGeometry_SB(FFT_INVERSE, &dev_out, &dev_in, n);    
    hipMemcpy(in, dev_in, n * sizeof(cpx), hipMemcpyDeviceToHost);

    return fftResultAndFree(n, &dev_in, &dev_out, NULL, &in, &ref, &out) != 1;
}

__host__ double tsConstantGeometry_SB_Performance(const int n)
{
    double measures[NUM_PERFORMANCE];
    cpx *in, *ref, *out, *dev_in, *dev_out;
    fftMalloc(n, &dev_in, &dev_out, NULL, &in, &ref, &out);

    hipMemcpy(dev_in, in, n * sizeof(cpx), hipMemcpyHostToDevice);
    for (int i = 0; i < NUM_PERFORMANCE; ++i) {
        startTimer();
        tsConstantGeometry_SB(FFT_FORWARD, &dev_in, &dev_out, n);
        measures[i] = stopTimer();
    }

    fftResultAndFree(n, &dev_in, &dev_out, NULL, &in, &ref, &out);
    return avg(measures, NUM_PERFORMANCE);
}

__host__ void tsConstantGeometry_SB(fftDirection dir, cpx **dev_in, cpx **dev_out, const int n)
{
    int threadsPerBlock, numBlocks;
    const float w_angle = dir * (M_2_PI / n);
    const cpx scale = make_hipFloatComplex((dir == FFT_FORWARD ? 1.f : 1.f / n), 0.f);   
    const int depth = log2_32(n);
    hipDeviceSetCacheConfig(hipFuncCachePreferShared);
    setBlocksAndThreads(&numBlocks, &threadsPerBlock, n);
#ifdef PRECALC_TWIDDLE
    int sharedMem = sizeof(cpx) * (n + n / 2);
    sharedMem = sharedMem > SHARED_MEM_SIZE ? SHARED_MEM_SIZE : sharedMem;
    _kernelCGSB KERNEL_ARGS3(numBlocks, threadsPerBlock, sharedMem)(*dev_in, *dev_out, w_angle, scale, depth, 32 - depth, n / 2);
#else
    int sharedMem = sizeof(cpx) * n;
    sharedMem = sharedMem > SHARED_MEM_SIZE ? SHARED_MEM_SIZE : sharedMem;
    _kernelCGSB48K KERNEL_ARGS3(numBlocks, threadsPerBlock, sharedMem)(*dev_in, *dev_out, w_angle, scale, depth, 32 - depth, n / 2);
#endif
    hipDeviceSynchronize();
}

__global__ void _kernelCGSB(cpx *in, cpx *out, const float angle, const cpx scale, int depth, const unsigned int lead, const int n2)
{
    extern __shared__ cpx mem[]; // sizeof(cpx) * (n + n + n/n)  
    cpx in_lower, in_upper;
    int tid = (blockIdx.x * blockDim.x + threadIdx.x);
    int in_low = n2 + tid;
    int in_high = n2 + in_low;
    int i = n2 + tid * 2;
    int ii = i + 1;

    /* Twiddle factors */
    SIN_COS_F(angle * tid, &mem[tid].y, &mem[tid].x);

    /* Move Global to Shared */
    globalToShared(2 * n2, tid, lead, mem, in);

    // Sync, as long as one block, not needed(?)
    SYNC_THREADS;

    /* Run FFT algorithm */
    for (int steps = 0; steps < depth; ++steps) {
        in_lower = mem[in_low];
        in_upper = mem[in_high];
        // Sync, as long as one block, not needed(?)
        SYNC_THREADS;
        mem[i] = hipCaddf(in_lower, in_upper);
        mem[ii] = hipCmulf(mem[(tid & (0xffffffff << steps))], hipCsubf(in_lower, in_upper));
        // A = B*(C-D) = B*C - B*D
        // Look for some single precision intrisics ;-)
        
        // Sync, as long as one block, not needed(?)
        SYNC_THREADS;
    }

    /* Move Shared to Global (index bit-reversed) */
    sharedToGlobal(n2 * 2, tid, scale, lead, mem, out);
}

__global__ void _kernelCGSB48K(cpx *in, cpx *out, const float angle, const cpx scale, const int depth, const unsigned int lead, const int n2)
{
    extern __shared__ cpx mem[]; 
    cpx w, in_lower, in_upper;
    int tid = (blockIdx.x * blockDim.x + threadIdx.x);
    int in_low = n2 + tid;
    int in_high = n2 + in_low;
    int i = tid * 2;
    int ii = i + 1;

    /* Move (bit-reversed?) Global to Shared */
    globalToShared(n2 * 2, tid, lead, mem, in);

    // Sync, as long as one block, not needed(?)
    SYNC_THREADS;

    /* Run FFT algorithm */
    for (int steps = 0; steps < depth; ++steps) {
        in_lower = mem[in_low];
        in_upper = mem[in_high];
        // Sync, as long as one block, not needed(?)
        SIN_COS_F(angle * ((tid & (0xffffffff << steps))), &w.y, &w.x);
        SYNC_THREADS;
        mem[i] = hipCaddf(in_lower, in_upper);
        mem[ii] = hipCmulf(w, hipCsubf(in_lower, in_upper));
        // A = B*(C-D) = B*C - B*D
        // Look for some single precision intrisics ;-)

        // Sync, as long as one block, not needed(?)
        SYNC_THREADS;
    }

    /* Move Shared to Global (index bit-reversed) */
    sharedToGlobal(n2 * 2, tid, scale, lead, mem, out);
}