#include "hip/hip_runtime.h"
#include <stdio.h>
#include <>

#include "tsHelper.cuh"
#include "math.h"

/* Doubtful this works... */
__host__ hipTextureObject_t specifyTexture(cpx *dev_W)
{
    // Specify texture
    struct hipResourceDesc resDesc;
    hipMemset(&resDesc, 0, sizeof(resDesc));
    resDesc.resType = hipResourceTypeArray;
    resDesc.res.linear.devPtr = dev_W;
    //resDesc.res.array.array = cuArray; 

    // Specify texture object parameters 
    struct hipTextureDesc texDesc;
    hipMemset(&texDesc, 0, sizeof(texDesc));
    texDesc.addressMode[0] = hipAddressModeWrap;
    texDesc.addressMode[1] = hipAddressModeWrap;
    texDesc.filterMode = hipFilterModeLinear;
    texDesc.readMode = hipReadModeElementType;
    texDesc.normalizedCoords = 1;

    // Create texture object 
    hipTextureObject_t texObj = 0;
    hipCreateTextureObject(&texObj, &resDesc, &texDesc, NULL);

    return texObj;
}

__host__ void setBlocksAndThreads(int *numBlocks, int *threadsPerBlock, const int size)
{
    if (size > MAX_BLOCK_SIZE) {
        *numBlocks = size / MAX_BLOCK_SIZE;
        *threadsPerBlock = MAX_BLOCK_SIZE;
    }
    else {
        *numBlocks = 1;
        *threadsPerBlock = size;
    }
}

__global__ void twiddle_factors(cpx *W, const float angle, const int n)
{
    int i = (blockIdx.x * blockDim.x + threadIdx.x);
    SIN_COS_F(angle * i, &W[i].y, &W[i].x);
}

__global__ void bit_reverse(cpx *in, cpx *out, const float scale, const int lead)
{
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int p = BIT_REVERSE(i, lead);
    out[p].x = in[i].x * scale;
    out[p].y = in[i].y * scale;
}

__global__ void bit_reverse(cpx *x, const float dir, const int lead, const int n)
{
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int p = BIT_REVERSE(i, lead);
    cpx tmp;
    if (i < p) {
        tmp = x[i];
        x[i] = x[p];
        x[p] = tmp;
    }
    if (dir > 0) {
        x[i].x = x[i].x / (float)n;
        x[i].y = x[i].y / (float)n;
    }
}