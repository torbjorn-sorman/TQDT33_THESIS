#include "hip/hip_runtime.h"
#include <stdio.h>
#include <>

#include "math.h"
#include "tsRegular.cuh"
#include "tsHelper.cuh"
#include "tsTest.cuh"

__global__ void _tsRegular_body(cpx *in, cpx *out, cpx *W, const int dist, const int dist2, const int n2);

__host__ int tsRegular_Validate(const size_t n)
{
    cpx *in, *ref, *out, *dev_in, *dev_out, *dev_W;
    fftMalloc(n, &dev_in, &dev_out, &dev_W, &in, &ref, &out);

    hipMemcpy(dev_in, in, n * sizeof(cpx), hipMemcpyHostToDevice);
    tsRegular(FFT_FORWARD, &dev_in, &dev_out, dev_W, n);
    tsRegular(FFT_INVERSE, &dev_out, &dev_in, dev_W, n);
    hipMemcpy(in, dev_in, n * sizeof(cpx), hipMemcpyDeviceToHost);

    return fftResultAndFree(n, &dev_in, &dev_out, &dev_W, &in, &ref, &out) != 1;
}

__host__ double tsRegular_Performance(const size_t n)
{
    double measures[NUM_PERFORMANCE];
    cpx *in, *ref, *out, *dev_in, *dev_out, *dev_W;
    fftMalloc(n, &dev_in, &dev_out, &dev_W, &in, &ref, &out);

    hipMemcpy(dev_in, in, n * sizeof(cpx), hipMemcpyHostToDevice);
    for (int i = 0; i < NUM_PERFORMANCE; ++i) {
        startTimer();
        tsRegular(FFT_FORWARD, &dev_in, &dev_out, dev_W, n);        
        measures[i] = stopTimer();
    }

    fftResultAndFree(n, &dev_in, &dev_out, &dev_W, &in, &ref, &out);
    return avg(measures, NUM_PERFORMANCE);
}

__host__ void tsRegular(fftDirection dir, cpx **dev_in, cpx **dev_out, cpx *dev_W, const int n)
{
    int dist, dist2, threadsPerBlock, numBlocks;
    const int n2 = (n / 2);
    const float w_angle = dir * (M_2_PI / n);
    const float scale = dir == FFT_FORWARD ? 1.f : 1.f / n;
    dist2 = n;
    dist = n2;

    setBlocksAndThreads(&numBlocks, &threadsPerBlock, n2);
    //twiddle_factors KERNEL_ARGS2(numBlocks, threadsPerBlock)(dev_W, w_angle, n);
    //hipDeviceSynchronize();
    
    //_tsRegular_body KERNEL_ARGS2(numBlocks, threadsPerBlock)(*dev_in, *dev_out, dev_W, dist, dist2, n);
    //hipDeviceSynchronize();
    while ((dist2 = dist) > 1) {
        dist = dist >> 1;
        //_tsRegular_body KERNEL_ARGS2(numBlocks, threadsPerBlock)(*dev_out, *dev_out, dev_W, dist, dist2, n);
        //hipDeviceSynchronize();
    }
    
    setBlocksAndThreads(&numBlocks, &threadsPerBlock, n);
    //bit_reverse KERNEL_ARGS2(numBlocks, threadsPerBlock)(*dev_out, *dev_in, scale, 32 - log2_32(n));
    //hipDeviceSynchronize();
    swap(dev_in, dev_out);
}

__global__ void _tsRegular_body(cpx *in, cpx *out, cpx *W, const int dist, const int dist2, const int n2)
{
}

/*

__inline void _fft_inner_body(cpx *in, cpx *out, const cpx *W, const int lower, const int upper, const int dist, const int mul)
{
    int u, p;
    float tmp_r, tmp_i;
    for (int l = lower; l < upper; ++l) {
        u = l + dist;
        p = (l - lower) * mul;
        tmp_r = in[l].r - in[u].r;
        tmp_i = in[l].i - in[u].i;
        out[l].r = in[l].r + in[u].r;
        out[l].i = in[l].i + in[u].i;
        out[u].r = (W[p].r * tmp_r) - (W[p].i * tmp_i);
        out[u].i = (W[p].i * tmp_r) + (W[p].r * tmp_i);
    }
}

__inline void _fft_body(cpx *in, cpx *out, cpx *W, int dist, int dist2, const int n_threads, const int n)
{
    const int count = n / dist2;
#ifdef _OPENMP        
    if (count >= n_threads) {
#pragma omp parallel for schedule(static)              
        for (int lower = 0; lower < n; lower += dist2) {
            _fft_inner_body(in, out, W, lower, dist + lower, dist, count);
        }
    }
    else
    {
        int u, p, upper;
        float tmp_r, tmp_i;
        for (int lower = 0; lower < n; lower += dist2) {
            upper = dist + lower;
#pragma omp parallel for schedule(static) private(u, p, tmp_r, tmp_i)
            for (int l = lower; l < upper; ++l) {
                u = l + dist;
                p = (l - lower) * count;
                tmp_r = in[l].r - in[u].r;
                tmp_i = in[l].i - in[u].i;
                out[l].r = in[l].r + in[u].r;
                out[l].i = in[l].i + in[u].i;
                out[u].r = (W[p].r * tmp_r) - (W[p].i * tmp_i);
                out[u].i = (W[p].i * tmp_r) + (W[p].r * tmp_i);
            }
        }
    }
#else
    for (int lower = 0; lower < n; lower += dist2) {
        fft_inner_body(in, out, W, lower, dist + lower, dist, count);
    }
#endif
}

*/