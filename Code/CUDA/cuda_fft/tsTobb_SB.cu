#include "hip/hip_runtime.h"
#include <stdio.h>
#include <>
#include <math.h>

#include "tsTobb_SB.cuh"
#include "tsHelper.cuh"
#include "tsTest.cuh"

__global__ void _kernelTSB(cpx *in, cpx *out, const int depth, const float angle, const cpx scale, const int n);
__global__ void _kernelTSB48K(cpx *in, cpx *out, const int depth, const float angle, const cpx scale, const int n);

__host__ int tsTobb_SB_Validate(const int n)
{
    cpx *in, *ref, *out, *dev_in, *dev_out;
    fftMalloc(n, &dev_in, &dev_out, NULL, &in, &ref, &out);

    hipMemcpy(dev_in, in, n * sizeof(cpx), hipMemcpyHostToDevice);
    tsTobb_SB(FFT_FORWARD, &dev_in, &dev_out, n);
    tsTobb_SB(FFT_INVERSE, &dev_out, &dev_in, n);
    hipMemcpy(in, dev_in, n * sizeof(cpx), hipMemcpyDeviceToHost);

    return fftResultAndFree(n, &dev_in, &dev_out, NULL, &in, &ref, &out) != 1;
}

__host__ double tsTobb_SB_Performance(const int n)
{
    double measures[NUM_PERFORMANCE];
    cpx *in, *ref, *out, *dev_in, *dev_out;
    fftMalloc(n, &dev_in, &dev_out, NULL, &in, &ref, &out);

    hipMemcpy(dev_in, in, n * sizeof(cpx), hipMemcpyHostToDevice);
    for (int i = 0; i < NUM_PERFORMANCE; ++i) {
        startTimer();
        tsTobb_SB(FFT_FORWARD, &dev_in, &dev_out, n);
        measures[i] = stopTimer();
    }

    fftResultAndFree(n, &dev_in, &dev_out, NULL, &in, &ref, &out);
    return avg(measures, NUM_PERFORMANCE);
}

__host__ void tsTobb_SB(fftDirection dir, cpx **dev_in, cpx **dev_out, const int n)
{
    int threadsPerBlock, numBlocks;
    const float w_angle = dir * (M_2_PI / n);
    const cpx scale = make_hipFloatComplex((dir == FFT_FORWARD ? 1.f : 1.f / n), 0.f);
    set_block_and_threads(&numBlocks, &threadsPerBlock, n / 2);
#ifdef PRECALC_TWIDDLE
    _kernelTSB KERNEL_ARGS3(numBlocks, threadsPerBlock, sizeof(cpx) * (n + n / 2))(*dev_in, *dev_out, log2_32(n), w_angle, scale, n);
#else
    _kernelTSB48K KERNEL_ARGS3(numBlocks, threadsPerBlock, sizeof(cpx) * n)(*dev_in, *dev_out, log2_32(n), w_angle, scale, n);
#endif
    hipDeviceSynchronize();
}

__global__ void _kernelTSB(cpx *in, cpx *out, const int depth, const float angle, const cpx scale, const int n)
{
    extern __shared__ cpx shared[];
    int tid = (blockIdx.x * blockDim.x + threadIdx.x);
    int offset = (n >> 1);
    int bit = depth;
    int dist = n;
    int lower;
    cpx w, in_lower, in_upper;

    /* Twiddle factors */
    SIN_COS_F(angle * tid, &shared[tid].y, &shared[tid].x);

    /* Move (bit-reversed?) Global to Shared */
    globalToShared(tid + offset, tid + n, offset, 32 - depth, shared, in);
    SYNC_THREADS;

    /* Run FFT algorithm */
    for (int steps = 0; steps < depth; ++steps) {
        --bit;
        dist /= 2;
        lower = offset + tid + (tid & (0xFFFFFFFF << bit));
        in_lower = shared[lower];
        in_upper = shared[lower + dist];
        SYNC_THREADS;
        w = shared[(tid << steps) & ((dist - 1) << steps)];
        cpx_add_sub_mul(&(shared[lower]), &(shared[lower + dist]), in_lower, in_upper, w);
        SYNC_THREADS;
    }

    /* Move (bit-reversed?) Shared to Global */
    sharedToGlobal(tid + offset, tid + n, offset, scale, 32 - depth, shared, out);
}

__global__ void _kernelTSB48K(cpx *in, cpx *out, const int depth, const float angle, const cpx scale, const int n)
{
    extern __shared__ cpx shared[];
    const int tid = (blockIdx.x * blockDim.x + threadIdx.x);
    const int lead = 32 - depth;
    int bit = depth;
    int dist = n;
    int lower;
    cpx w, in_lower, in_upper;

    /* Move Global to Shared */
    globalToShared(tid, tid + (n >> 1), 0, lead, shared, in);
    SYNC_THREADS;

    /* Run FFT algorithm */
    for (int steps = 0; steps < depth; ++steps) {   
        --bit;
        dist /= 2;
        lower = tid + (tid & (0xFFFFFFFF << bit));
        in_lower = shared[lower];
        in_upper = shared[lower + dist];
        SIN_COS_F(angle * ((tid << steps) & ((dist - 1) << steps)), &w.y, &w.x);
        SYNC_THREADS;
        cpx_add_sub_mul(&(shared[lower]), &(shared[lower + dist]), in_lower, in_upper, w);
        SYNC_THREADS;
    }

    /* Move Shared to Global */
    sharedToGlobal(tid, tid + (n >> 1), 0, scale, lead, shared, out);
}