#include "hip/hip_runtime.h"
#include <stdio.h>
#include <>
#include "math.h"

#include "tsTobb_SB.cuh"
#include "tsHelper.cuh"
#include "tsTest.cuh"

__global__ void _tsTobb_SB_body(cpx *in, cpx *out, const int depth, const float angle, const cpx scale, const int n);

__host__ int tsTobb_SB_Validate(const size_t n)
{
    int result;
    cpx *in, *ref, *out, *dev_in, *dev_out;
    fftMalloc(n, &dev_in, &dev_out, NULL, &in, &ref, &out);

    hipMemcpy(dev_in, in, n * sizeof(cpx), hipMemcpyHostToDevice);
    tsTobb_SB(FFT_FORWARD, &dev_in, &dev_out, n);
    tsTobb_SB(FFT_INVERSE, &dev_out, &dev_in, n);
    hipMemcpy(in, dev_in, n * sizeof(cpx), hipMemcpyDeviceToHost);

    return fftResultAndFree(n, &dev_in, &dev_out, NULL, &in, &ref, &out) != 1;
}

__host__ double tsTobb_SB_Performance(const size_t n)
{
    double measures[NUM_PERFORMANCE];
    cpx *in, *ref, *out, *dev_in, *dev_out;
    fftMalloc(n, &dev_in, &dev_out, NULL, &in, &ref, &out);

    hipMemcpy(dev_in, in, n * sizeof(cpx), hipMemcpyHostToDevice);
    for (int i = 0; i < NUM_PERFORMANCE; ++i) {
        startTimer();
        tsTobb_SB(FFT_FORWARD, &dev_in, &dev_out, n);
        measures[i] = stopTimer();
    }

    fftResultAndFree(n, &dev_in, &dev_out, NULL, &in, &ref, &out);
    return avg(measures, NUM_PERFORMANCE);
}

__host__ void tsTobb_SB(fftDirection dir, cpx **dev_in, cpx **dev_out, const int n)
{
    int threadsPerBlock, numBlocks;
    const float w_angle = dir * (M_2_PI / n);
    const cpx scale = make_hipFloatComplex((dir == FFT_FORWARD ? 1.f : 1.f / n), 0.f);
    const int sharedMem = min(sizeof(cpx) * (n + n / 2), SHARED_MEM_SIZE);
    hipDeviceSetCacheConfig(hipFuncCachePreferShared);
    
    _tsTobb_SB_body KERNEL_ARGS3(numBlocks, threadsPerBlock, sharedMem)(*dev_in, *dev_out, log2_32(n), w_angle, scale, n);
    hipDeviceSynchronize();
}

__global__ void _tsTobb_SB_body(cpx *in, cpx *out, const int depth, const float angle, const cpx scale, const int n)
{
    extern __shared__ cpx mem[];
    int tid = (blockIdx.x * blockDim.x + threadIdx.x);
    int n2 = n / 2;
    int bit = depth;
    int dist = n;
    int lower;
    cpx in_lower, in_upper;

    /*
    Twiddle factors
    */
    SIN_COS_F(angle * tid, &mem[tid].y, &mem[tid].x);

    /*
    Move (bit-reversed?) Global to Shared
    */
    mem[n2 + tid * 2] = in[tid * 2];
    mem[n2 + tid * 2 + 1] = in[tid * 2 + 1];

    // Sync, as long as one block, not needed(?)
    SYNC_THREADS;

    /*
    Run FFT algorithm
    */
    for (int steps = 0; steps < depth; ++steps) {
        --bit;
        dist /= 2;
        lower = tid + (tid & (0xFFFFFFFF << bit));
        in_lower = mem[lower];
        in_upper = mem[lower + dist];        
        mem[lower] = hipCaddf(in_lower, in_upper);
        mem[lower + dist] = hipCmulf(mem[(tid << steps) & ((dist - 1) << steps)], hipCsubf(in_lower, in_upper));
        // Sync, as long as one block, not needed(?)
        SYNC_THREADS;
    }

    // Sync, as long as one block, not needed(?)
    SYNC_THREADS;

    /*
    Move (bit-reversed?) Shared to Global
    */    
    out[tid * 2] = hipCmulf(mem[BIT_REVERSE(n2 + tid * 2, lead)], scale);
    out[tid * 2 + 1] = hipCmulf(mem[BIT_REVERSE(n2 + tid * 2 + 1, lead)], scale);
}