#include "hip/hip_runtime.h"
#include <stdio.h>
#include <>
#include <math.h>

#include "tsTobb_SB.cuh"
#include "tsHelper.cuh"
#include "tsTest.cuh"

__global__ void _kernelTSB(cpx *in, cpx *out, const int depth, const float angle, const cpx scale, const int n);
__global__ void _kernelTSB48K(cpx *in, cpx *out, const int depth, const float angle, const cpx scale, const int n);

__host__ int tsTobb_SB_Validate(const int n)
{
    cpx *in, *ref, *out, *dev_in, *dev_out;
    fftMalloc(n, &dev_in, &dev_out, NULL, &in, &ref, &out);

    hipMemcpy(dev_in, in, n * sizeof(cpx), hipMemcpyHostToDevice);
    tsTobb_SB(FFT_FORWARD, &dev_in, &dev_out, n);
    tsTobb_SB(FFT_INVERSE, &dev_out, &dev_in, n);
    hipMemcpy(in, dev_in, n * sizeof(cpx), hipMemcpyDeviceToHost);

    return fftResultAndFree(n, &dev_in, &dev_out, NULL, &in, &ref, &out) != 1;
}

__host__ double tsTobb_SB_Performance(const int n)
{
    double measures[NUM_PERFORMANCE];
    cpx *in, *ref, *out, *dev_in, *dev_out;
    fftMalloc(n, &dev_in, &dev_out, NULL, &in, &ref, &out);

    hipMemcpy(dev_in, in, n * sizeof(cpx), hipMemcpyHostToDevice);
    for (int i = 0; i < NUM_PERFORMANCE; ++i) {
        startTimer();
        tsTobb_SB(FFT_FORWARD, &dev_in, &dev_out, n);
        measures[i] = stopTimer();
    }

    fftResultAndFree(n, &dev_in, &dev_out, NULL, &in, &ref, &out);
    return avg(measures, NUM_PERFORMANCE);
}

__host__ void tsTobb_SB(fftDirection dir, cpx **dev_in, cpx **dev_out, const int n)
{
    int threadsPerBlock, numBlocks;
    const float w_angle = dir * (M_2_PI / n);
    const cpx scale = make_hipFloatComplex((dir == FFT_FORWARD ? 1.f : 1.f / n), 0.f);
    hipDeviceSetCacheConfig(hipFuncCachePreferShared);
    setBlocksAndThreads(&numBlocks, &threadsPerBlock, n);
#ifdef PRECALC_TWIDDLE
    int sharedMem = sizeof(cpx) * (n + n / 2);
    sharedMem = sharedMem > SHARED_MEM_SIZE ? SHARED_MEM_SIZE : sharedMem;
    _kernelTSB KERNEL_ARGS3(numBlocks, threadsPerBlock, sharedMem)(*dev_in, *dev_out, log2_32(n), w_angle, scale, n);
#else
    int sharedMem = sizeof(cpx) * n;
    sharedMem = sharedMem > SHARED_MEM_SIZE ? SHARED_MEM_SIZE : sharedMem;
    _kernelTSB48K KERNEL_ARGS3(numBlocks, threadsPerBlock, sharedMem)(*dev_in, *dev_out, log2_32(n), w_angle, scale, n);
#endif
    hipDeviceSynchronize();
}

__global__ void _kernelTSB(cpx *in, cpx *out, const int depth, const float angle, const cpx scale, const int n)
{
    extern __shared__ cpx mem[];
    int tid = (blockIdx.x * blockDim.x + threadIdx.x);
    int offset = n / 2;
    int bit = depth;
    int dist = n;
    int lower;
    cpx in_lower, in_upper;

    /* Twiddle factors */
    SIN_COS_F(angle * tid, &mem[tid].y, &mem[tid].x);

    /* Move (bit-reversed?) Global to Shared */    
    globalToShared(n, tid, offset, mem, in);

    // Sync, as long as one block, not needed(?)
    SYNC_THREADS;

    /* Run FFT algorithm */
    for (int steps = 0; steps < depth; ++steps) {
        --bit;
        dist /= 2;
        lower = offset + tid + (tid & (0xFFFFFFFF << bit));
        in_lower = mem[lower];
        in_upper = mem[lower + dist];
        mem[lower] = hipCaddf(in_lower, in_upper);
        mem[lower + dist] = hipCmulf(mem[(tid << steps) & ((dist - 1) << steps)], hipCsubf(in_lower, in_upper));
        // Sync, as long as one block, not needed(?)
        SYNC_THREADS;
    }

    /* Move (bit-reversed?) Shared to Global */
    sharedToGlobal(n, tid, scale, 32 - depth, mem, out);
}

__global__ void _kernelTSB48K(cpx *in, cpx *out, const int depth, const float angle, const cpx scale, const int n)
{
    extern __shared__ cpx mem[];
    int tid = (blockIdx.x * blockDim.x + threadIdx.x);
    int bit = depth;
    int dist = n;
    int lower;
    cpx w, in_lower, in_upper;

    /* Move (bit-reversed?) Global to Shared */
    globalToShared(n, tid, 32 - depth, mem, in);

    // Sync, as long as one block, not needed(?)
    SYNC_THREADS;

    /* Run FFT algorithm */
    for (int steps = 0; steps < depth; ++steps) {
        --bit;
        dist /= 2;
        lower = tid + (tid & (0xFFFFFFFF << bit));
        in_lower = mem[lower];
        in_upper = mem[lower + dist];
        SIN_COS_F(angle * ((tid << steps) & ((dist - 1) << steps)), &w.y, &w.x);
        mem[lower] = hipCaddf(in_lower, in_upper);
        mem[lower + dist] = hipCmulf(w, hipCsubf(in_lower, in_upper));
        // Sync, as long as one block, not needed(?)
        SYNC_THREADS;
    }

    /* Move (bit-reversed?) Shared to Global */
    sharedToGlobal(n, tid, scale, 32 - depth, mem, out);
}